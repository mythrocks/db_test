#include "hip/hip_runtime.h"
#include "db_test.cuh"

#if 0    // copy_if_else
//namespace db_test {

using namespace cudf;
using namespace std;
using namespace rmm;
using namespace rmm::mr;

namespace {

using namespace cudf; 

/* --------------------------------------------------------------------------*/
/**
* @brief Functor called by the `type_dispatcher` in order to perform a copy if/else
*        using a filter function to select from lhs/rhs columns.
*/
/* ----------------------------------------------------------------------------*/
struct copy_if_else_functor {
   template <typename T, typename Filter>
   void operator()(  Filter filter,
                     column_view const& lhs,
                     column_view const& rhs,
                     mutable_column_view& out,
                     hipStream_t stream)
   {
      auto begin  = thrust::make_zip_iterator(thrust::make_tuple( thrust::make_counting_iterator(0),
                                                                  lhs.begin<T>(),
                                                                  rhs.begin<T>()));

      auto end  = thrust::make_zip_iterator(thrust::make_tuple(thrust::make_counting_iterator(lhs.size()),
                                                               lhs.end<T>(),
                                                               rhs.end<T>()));
      
      thrust::transform(rmm::exec_policy(stream)->on(stream), begin, end, out.begin<T>(),
                        [filter] __device__ (thrust::tuple<size_type, T, T> i)
                        {
                           return filter(thrust::get<0>(i)) ? thrust::get<1>(i) : thrust::get<2>(i);
                        });
   } 
};

#endif   // copy if else

#if 0
void copy_if_else_test()
{         
   /*
   using T = int;

   // short one. < 1 warp/bitmask length
   int num_els = 4;

   bool mask[]    = { 1, 0, 0, 0 };
   bool_wrapper mask_w(mask, mask + num_els);

   T lhs[]        = { 5, 5, 5, 5 }; 
   bool lhs_v[]   = { 1, 1, 1, 1 };
   wrapper<T> lhs_w(lhs, lhs + num_els, lhs_v);

   T rhs[]        = { 6, 6, 6, 6 };
   bool rhs_v[]   = { 1, 1, 1, 1 };
   wrapper<T> rhs_w(rhs, rhs + num_els, rhs_v);
   
   T expected[]   = { 5, 6, 6, 6 };
   // bool exp_v[]   = { 1, 1, 1, 1 };
   wrapper<T> expected_w(expected, expected + num_els);

   auto out = cudf::experimental::copy_if_else(lhs_w, rhs_w, mask_w);
   print_column(*out);
   cudf::test::expect_columns_equal(out->view(), expected_w);
   */
   auto valids = cudf::test::make_counting_transform_iterator(0, [](auto i) { return i % 2 == 0 ? true : false; });

   std::vector<const char*> h_string1{ "eee" };   
   cudf::string_scalar strings1{h_string1[0]};
   std::vector<const char*> h_strings2{ "zz",  "", "yyy", "w", "ééé", "ooo" };
   cudf::test::strings_column_wrapper strings2( h_strings2.begin(), h_strings2.end(), valids );   

   bool mask[] = { 1, 0, 1, 0, 1, 0 };
   bool_wrapper mask_w(mask, mask + 6);  
      
   auto results = cudf::experimental::copy_if_else(strings1, strings2, mask_w);
      
   std::vector<const char*> h_expected;
   for( cudf::size_type idx=0; idx < static_cast<cudf::size_type>(h_strings2.size()); ++idx )
   {
      if( mask[idx] ){
         h_expected.push_back( h_string1[0] );
      } else {
         h_expected.push_back( h_strings2[idx] );
      }
   }
   cudf::test::strings_column_wrapper expected( h_expected.begin(), h_expected.end(), valids);   
   print_column(*results);
   cudf::test::expect_columns_equal(*results,expected);;

   

   /*
   // make sure we span at least 2 warps      
   int num_els = 64;
   
   bool mask[]    = { 1, 0, 1, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
                     1, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 };   
   bool_wrapper mask_w(mask, mask + num_els);

   T lhs[]        = { 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 
                     5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5 };   
   bool lvalid[] = { 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
                     0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0 };
   wrapper<T> lhs_w(lhs, lhs + num_els, lvalid);

   T rhs[]        = { 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6,
                     6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6 };
   bool rvalid[] = { 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
                     1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 };
   wrapper<T> rhs_w(rhs, rhs + num_els, rvalid);
   
   T expected[]   = { 5, 6, 5, 6, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 6, 6, 6, 6, 5, 5, 5, 5, 5, 5, 5, 5, 5, 
                     5, 6, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5 };   
   bool valid_e[] = { 0, 1, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
                     0, 1, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0 };
   wrapper<T> expected_w(expected, expected + num_els, valid_e);
   */      
}
#endif

#if 0 // copy_if_else
/*
std::unique_ptr<column> _copy_if_else( cudf::scalar const& lhs, column_view const& rhs, column_view const& boolean_mask,
                                      rmm::mr::device_memory_resource *mr = rmm::mr::get_default_resource(),
                                      hipStream_t stream = 0)
{
   CUDF_EXPECTS(lhs.type() == rhs.type(), "Both columns must be of the same type");   
   CUDF_EXPECTS(not boolean_mask.has_nulls(), "Boolean mask must not contain null values.");
   CUDF_EXPECTS(boolean_mask.type() == data_type(BOOL8), "Boolean mask column must be of type BOOL8");   
   CUDF_EXPECTS(boolean_mask.size() == rhs.size(), "Boolean mask column must be the same size as lhs and rhs columns");   

   auto bool_mask_device_p = column_device_view::create(boolean_mask);
   column_device_view bool_mask_device = *bool_mask_device_p;
   auto filter = [bool_mask_device] __device__ (cudf::size_type i) { return bool_mask_device.element<cudf::experimental::bool8>(i); };

   return cudf::experimental::detail::copy_if_else(lhs, rhs, filter, mr, stream);
}
*/

void copy_if_else_scalar_test()
{   
   using T = int;

   /*
   // short one. < 1 warp/bitmask length   
   int num_els = 5;

   bool mask[]    = { 1, 0, 0, 1, 0 };
   bool_wrapper mask_w(mask, mask + num_els);

   T lhs[]        = { 99, 5, 5, 99, 5 };
   bool lhs_v[]   = { 1, 1, 1, 1, 1 };  
   // wrapper<T> lhs_w(lhs, lhs + num_els, lhs_v);   
   cudf::numeric_scalar<T> lhs_w(88);   

   T rhs[]        = { 6, 6, 6, 6, 6 };  
   bool rhs_v[]   = { 1, 1, 1, 1, 0 };  
   // wrapper<T> rhs_w(rhs, rhs + num_els, rhs_v);
   // column_view rhs_c(rhs_w);
   cudf::numeric_scalar<T> rhs_w(77);   
      
   T expected[]        = { 99, 6, 6, 99, 6 };  
   bool expected_v[]   = { 1, 1, 1, 1, 0 };
   wrapper<T> expected_w(expected, expected + num_els, expected_v);   
      
   auto out = cudf::experimental::copy_if_else(lhs_w, rhs_w, mask_w);
   column_view out_v = out->view();
   print_column(out_v);
   cudf::test::expect_columns_equal(out->view(), expected_w);
   */
  /*
   int num_els = 4;

   bool mask[]    = { 1, 0, 0, 1 };
   bool_wrapper mask_w(mask, mask + num_els);

   cudf::numeric_scalar<T> lhs_w(5);

   cudf::numeric_scalar<T> rhs_w(6);
   
   T expected[]   = { 5, 6, 6, 5 };   
   wrapper<T> expected_w(expected, expected + num_els);

   auto out = cudf::experimental::copy_if_else(lhs_w, rhs_w, mask_w);
   column_view out_v = out->view();
   print_column(out_v);
   cudf::test::expect_columns_equal(out_v, expected_w);   
   */

   {
      int num_els = 4;

      bool mask[]    = { 1, 0, 1, 1 };
      bool_wrapper mask_w(mask, mask + num_els);

      T lhs[]        = { 5, 5, 5, 5 }; 
      bool lhs_m[]   = { 1, 1, 1, 0 };
      wrapper<T> lhs_w(lhs, lhs + num_els, lhs_m);

      T rhs[]        = { 6, 6, 6, 6 };
      bool rhs_m[]   = { 1, 0, 1, 1 };
      wrapper<T> rhs_w(rhs, rhs + num_els, rhs_m);      

      T expected[]   = { 5, 6, 5, 5 };
      bool exp_m[]   = { 1, 0, 1, 0 };
      wrapper<T> expected_w(expected, expected + num_els, exp_m);

      auto out = cudf::experimental::copy_if_else(lhs_w, rhs_w, mask_w);     
      column_view out_v = out->view();
      print_column(out_v);
      cudf::test::expect_columns_equal(out->view(), expected_w);  
   }
   
   {
      int num_els = 4;

      bool mask[]    = { 1, 0, 1, 1 };
      bool_wrapper mask_w(mask, mask + num_els);

      T lhs[]        = { 5, 5, 5, 5 }; 
      bool lhs_m[]   = { 1, 1, 1, 0 };
      wrapper<T> lhs_w(lhs, lhs + num_els, lhs_m);

      T rhs[]        = { 6, 6, 6, 6 };
      wrapper<T> rhs_w(rhs, rhs + num_els);      

      T expected[]   = { 5, 6, 5, 5 };
      bool exp_m[]   = { 1, 1, 1, 0 };
      wrapper<T> expected_w(expected, expected + num_els, exp_m);

      auto out = cudf::experimental::copy_if_else(lhs_w, rhs_w, mask_w);     
      column_view out_v = out->view();
      print_column(out_v);
      cudf::test::expect_columns_equal(out->view(), expected_w);  
   }
   {
      int num_els = 4;

      bool mask[]    = { 1, 0, 1, 1 };
      bool_wrapper mask_w(mask, mask + num_els);

      T lhs[]        = { 5, 5, 5, 5 };       
      wrapper<T> lhs_w(lhs, lhs + num_els);

      T rhs[]        = { 6, 6, 6, 6 };
      bool rhs_m[]   = { 1, 0, 1, 1 };
      wrapper<T> rhs_w(rhs, rhs + num_els, rhs_m);      

      T expected[]   = { 5, 6, 5, 5 };
      bool exp_m[]   = { 1, 0, 1, 1 };
      wrapper<T> expected_w(expected, expected + num_els, exp_m);

      auto out = cudf::experimental::copy_if_else(lhs_w, rhs_w, mask_w);     
      column_view out_v = out->view();
      print_column(out_v);
      cudf::test::expect_columns_equal(out->view(), expected_w);  
   }
   
   {
      int num_els = 4;

      bool mask[]    = { 1, 0, 1, 1 };
      bool_wrapper mask_w(mask, mask + num_els);

      T lhs[]        = { 5, 5, 5, 5 };       
      wrapper<T> lhs_w(lhs, lhs + num_els);

      T rhs[]        = { 6, 6, 6, 6 };
      wrapper<T> rhs_w(rhs, rhs + num_els);      

      T expected[]   = { 5, 6, 5, 5 };      
      wrapper<T> expected_w(expected, expected + num_els);

      auto out = cudf::experimental::copy_if_else(lhs_w, rhs_w, mask_w);     
      column_view out_v = out->view();
      print_column(out_v);
      cudf::test::expect_columns_equal(out->view(), expected_w);  
   }
}

// #endif // copy_if_else
#endif


#if 0

/**
 * @brief Copies contents of `in` to `out`.  Copies validity if present
 * but does not compute null count.
 *  
 * @param in column_view to copy from
 * @param out mutable_column_view to copy to.
 */
template <size_type block_size, typename T, bool has_validity>
__launch_bounds__(block_size)
__global__
void _copy_in_place_kernel( column_device_view const in,
                           size_type validity_size,
                           mutable_column_device_view out,
                           T val_subtract)
{
   const size_type tid = threadIdx.x + blockIdx.x * block_size;
   const int warp_id = tid / cudf::experimental::detail::warp_size;
   const size_type warps_per_grid = gridDim.x * block_size / cudf::experimental::detail::warp_size;      

   // begin/end indices for the column data
   size_type begin = 0;      
   size_type end = in.size();
   //if(tid == 0){ printf("end : %d\n", end); }
   size_type validity_end = validity_size;
   //if(tid == 0){ printf("validity_end : %d\n", validity_end); }
   //printf("end : %d\n", end);
   //printf("validity_end : %d\n", validity_end);
   // warp indices.  since 1 warp == 32 threads == sizeof(bit_mask_t) * 8,
   // each warp will process one (32 bit) of the validity mask via
   // __ballot_sync()
   size_type warp_begin = cudf::word_index(begin);
   //if(tid == 0){ printf("warp_begin : %d\n", warp_begin); }
   size_type warp_end = cudf::word_index(end-1);      
   //if(tid == 0){ printf("warp_end : %d\n", warp_begin); }

   // lane id within the current warp   
   const int lane_id = threadIdx.x % cudf::experimental::detail::warp_size;
   
   // current warp.
   size_type warp_cur = warp_begin + warp_id;   
   size_type index = tid;
   while(warp_cur <= warp_end){
      bool validity_in_range = (index >= begin && index < validity_end);
      bool valid = true;      
      if(has_validity){         
         valid = validity_in_range && in.is_valid(index);
      }

      bool in_range = (index >= begin && index < end);
      if(in_range){
         //printf("copy : %d\n", index);
         out.element<T>(index) = in.element<T>(index) - val_subtract;
      }
      
      // update validity      
      if(has_validity && validity_in_range){
         // the final validity mask for this warp 
         int warp_mask = __ballot_sync(0xFFFF'FFFF, valid && validity_in_range);
         // only one guy in the warp needs to update the mask and count
         if(lane_id == 0){            
            out.set_mask_word(warp_cur, warp_mask);            
         }
      }            

      // next grid
      warp_cur += warps_per_grid;
      index += block_size * gridDim.x;
   }      
}

// align all column size allocations to this boundary so that all output column buffers
// start at that alignment.
static constexpr size_t split_align = 64;

struct column_split_info {
   size_t   data_size;     // size of the data
   size_t   validity_size; // validity vector size
   
   size_t   offsets_size;  // (strings only) size of offset column
   size_t   chars_size;    // (strings only) # of chars in the column
   size_t   chars_offset;  // (strings only) offset from head of chars data
};

/**
 * @brief Functor called by the `type_dispatcher` to incrementally compute total
 * memory buffer size needed to allocate a contiguous copy of all columns within
 * a source table. 
 */
struct _column_buffer_size_functor {   
   template <typename T, std::enable_if_t<not is_fixed_width<T>()>* = nullptr>
   size_t operator()(column_view const& c, column_split_info &split_info)
   {
      // this has already been precomputed in an earlier step      
      return split_info.data_size + split_info.validity_size + split_info.offsets_size;
   }

   template <typename T, std::enable_if_t<is_fixed_width<T>()>* = nullptr>
   size_t operator()(column_view const& c, column_split_info &split_info)
   {      
      split_info.data_size = cudf::util::round_up_safe(c.size() * sizeof(T), split_align);  
      split_info.validity_size = (c.nullable() ? cudf::bitmask_allocation_size_bytes(c.size(), split_align) : 0);
      return split_info.data_size + split_info.validity_size;
   }
};

/**
 * @brief Functor called by the `type_dispatcher` to copy a column into a contiguous
 * buffer of output memory. 
 * 
 * Used for copying each column in a source table into one contiguous buffer of memory.
 */
struct _column_copy_functor {
   template <typename T, std::enable_if_t<not is_fixed_width<T>()>* = nullptr>
   void operator()(column_view const& in, column_split_info const& split_info, char*& dst, std::vector<column_view>& out_cols)
   {            
      strings_column_view strings_c(in);      

      // outgoing pointers
      char* chars_buf = dst;
      bitmask_type* validity_buf = split_info.validity_size == 0 ? nullptr : reinterpret_cast<bitmask_type*>(dst + split_info.data_size);
      char* offsets_buf = dst + split_info.data_size + split_info.validity_size;

      // increment working buffer
      dst += (split_info.data_size + split_info.validity_size + split_info.offsets_size);

      // 2 kernel calls. 1 to copy offsets and validity, and another to copy chars
      
      column_view offsets_col = strings_c.offsets();
      print_column(offsets_col);
      column_view _chars_col = strings_c.chars();
      print_column(_chars_col);
      
      // copy offsets and validity
      mutable_column_view temp_offsets_and_validity{
                              offsets_col.type(), offsets_col.size(), offsets_buf,
                              validity_buf, validity_buf == nullptr ? UNKNOWN_NULL_COUNT : 0,
                              0 };
      {         
         // contruct a column which wraps the validity vector and the offsets from the child column. 
         // this is weird but it removes an extra kernel call. however, since the length of the offsets column
         // is always 1 greater than the # of strings, the validity vector will be short by 1. the kernel will have to
         // compensate for that. 
         CUDF_EXPECTS(in.size() == offsets_col.size()-1, "Expected offsets column to be the same size as parent");
         CUDF_EXPECTS(in.offset() == offsets_col.offset(), "Expected offsets column offset to be the same as parent");
         CUDF_EXPECTS(offsets_col.type() == cudf::data_type(INT32), "Expected offsets column type to be int32");
         column_view in_offsets_and_validity{
                                 offsets_col.type(), offsets_col.size(), offsets_col.head<int32_t>(),
                                 in.null_mask(), in.null_mask() == nullptr ? UNKNOWN_NULL_COUNT : 0,
                                 in.offset()};
         
         cudf::size_type num_els = cudf::util::round_up_safe(strings_c.offsets().size(), cudf::experimental::detail::warp_size);
         constexpr int block_size = 256;
         cudf::experimental::detail::grid_1d grid{num_els, block_size, 1};         
         if(in.nullable()){
            _copy_in_place_kernel<block_size, size_type, true><<<grid.num_blocks, block_size, 0, 0>>>(
                              *column_device_view::create(in_offsets_and_validity), 
                              in.size(),  // validity vector length
                              *mutable_column_device_view::create(temp_offsets_and_validity), split_info.chars_offset);
         } else {
            _copy_in_place_kernel<block_size, size_type, false><<<grid.num_blocks, block_size, 0, 0>>>(
                              *column_device_view::create(in_offsets_and_validity),
                              in.size(),  // validity vector length
                              *mutable_column_device_view::create(temp_offsets_and_validity), split_info.chars_offset);
         }
      }

      // get the chars column directly instead of calling .chars(), since .chars() will end up
      // doing gpu work we specifically want to avoid.
      column_view chars_col = in.child(strings_column_view::chars_column_index);

      // copy chars
      mutable_column_view out_chars{chars_col.type(), static_cast<size_type>(split_info.chars_size), chars_buf};
      {         
         CUDF_EXPECTS(!chars_col.nullable(), "Expected input chars column to not be nullable");
         CUDF_EXPECTS(chars_col.offset() == 0, "Expected input chars column to have an offset of 0");
         column_view in_chars{ chars_col.type(), static_cast<size_type>(split_info.chars_size), chars_col.data<char>() + split_info.chars_offset };
                                 
         cudf::size_type num_els = cudf::util::round_up_safe(static_cast<size_type>(split_info.chars_size), cudf::experimental::detail::warp_size);
         constexpr int block_size = 256;
         cudf::experimental::detail::grid_1d grid{num_els, block_size, 1};         
         _copy_in_place_kernel<block_size, char, false><<<grid.num_blocks, block_size, 0, 0>>>(
                           *column_device_view::create(in_chars),
                           split_info.chars_size,
                           *mutable_column_device_view::create(out_chars), 0);         
      }

      // construct output string column_view.  offsets and validity have been glued together so
      // we have to rearrange things a bit.      
      column_view out_offsets{strings_c.offsets().type(), strings_c.offsets().size(), offsets_buf};
      
      out_cols.push_back(column_view(in.type(), in.size(), nullptr,
                                     validity_buf, UNKNOWN_NULL_COUNT, 0,
                                     { out_offsets, out_chars }));
                                    
                                    // out_cols.push_back({});
   }

   template <typename T, std::enable_if_t<is_fixed_width<T>()>* = nullptr>
   void operator()(column_view const& in, column_split_info const& split_info, char*& dst, std::vector<column_view>& out_cols)
   {     
      // outgoing pointers
      char* data = dst;
      bitmask_type* validity = split_info.validity_size == 0 ? nullptr : reinterpret_cast<bitmask_type*>(dst + split_info.data_size);

      // increment working buffer
      dst += (split_info.data_size + split_info.validity_size);

      // custom copy kernel (which should probably just be an in-place copy() function in cudf.
      cudf::size_type num_els = cudf::util::round_up_safe(in.size(), cudf::experimental::detail::warp_size);
      constexpr int block_size = 256;
      cudf::experimental::detail::grid_1d grid{num_els, block_size, 1};
      
      // so there's a significant performance issue that comes up. our incoming column_view objects
      // are the result of a slice.  because of this, they have an UNKNOWN_NULL_COUNT.  because of that,
      // calling column_device_view::create() will cause a recompute of the count, which ends up being
      // extremely slow because a.) the typical use case here will involve huge numbers of calls and
      // b.) the count recompute involves tons of device allocs and memcopies.
      //
      // so to get around this, I am manually constructing a fake-ish view here where the null
      // count is arbitrarily bashed to 0.            
      //            
      // Remove this hack once rapidsai/cudf#3600 is fixed.
      column_view   in_wrapped{in.type(), in.size(), in.head<T>(), 
                               in.null_mask(), in.null_mask() == nullptr ? UNKNOWN_NULL_COUNT : 0,
                               in.offset() };
      mutable_column_view  mcv{in.type(), in.size(), data, 
                               validity, validity == nullptr ? UNKNOWN_NULL_COUNT : 0 };      
      if(in.nullable()){               
         _copy_in_place_kernel<block_size, T, true><<<grid.num_blocks, block_size, 0, 0>>>(
                           *column_device_view::create(in_wrapped), 
                           in.size(),
                           *mutable_column_device_view::create(mcv), 0);         
      } else {
         _copy_in_place_kernel<block_size, T, false><<<grid.num_blocks, block_size, 0, 0>>>(
                           *column_device_view::create(in_wrapped), 
                           in.size(),
                           *mutable_column_device_view::create(mcv), 0);
      }
      mcv.set_null_count(cudf::UNKNOWN_NULL_COUNT);                 

      out_cols.push_back(mcv);
   }
};

template <typename S>
__device__ inline S round_up_safe_nothrow(S number_to_round, S modulus) {
    auto remainder = number_to_round % modulus;
    if (remainder == 0) { return number_to_round; }
    auto rounded_up = number_to_round - remainder + modulus;    
    return rounded_up;
}

// Computes required allocation size of a bitmask
__device__ std::size_t bitmask_allocation_size_bytes_nothrow(size_type number_of_bits,
                                          std::size_t padding_boundary) {  
  auto necessary_bytes =
      cudf::util::div_rounding_up_safe<size_type>(number_of_bits, CHAR_BIT);

  auto padded_bytes =
      padding_boundary * cudf::util::div_rounding_up_safe<size_type>(
                             necessary_bytes, padding_boundary);
  return padded_bytes;
}

/**
 * @brief Creates a contiguous_split_result object which contains a deep-copy of the input
 * table_view into a single contiguous block of memory. 
 * 
 * The table_view contained within the contiguous_split_result will pass an expect_tables_equal()
 * call with the input table.  The memory referenced by the table_view and its internal column_views
 * is entirely contained in single block of memory.
 */
contiguous_split_result _alloc_and_copy(cudf::table_view const& t, rmm::mr::device_memory_resource* mr, hipStream_t stream)
{      
   /*
   // offsets for this column
   for(size_t idx=0; idx<(size_t)t.num_columns(); idx++){
      size_type whee[256] = {0};
      column_device_view ccco = offset_columns[idx];
      hipMemcpy(whee, ccco.data<int32_t>(), ccco.size() * sizeof(size_type), hipMemcpyDeviceToHost);
      
      printf("col %d (size : %d)\n", (size_type)idx, (size_type)ccco.size());
      for(size_type oidx=0; oidx<ccco.size(); oidx++){      
         printf("%d ", whee[oidx]);
      }
      printf("\n");
      printf("col.head : %d\n", whee[ccco.offset()]);
      printf("col.data : %d\n", whee[0]);
   }
   */

   /*
   // bring it back to the cpu
   thrust::host_vector<size_type> string_lengths = device_string_lengths;   
   printf("yay : ");
   for(size_t idx=0; idx<string_lengths.size(); idx++){
      printf("%d ", string_lengths[idx]);
   }
   printf("\n");

   int whee = 10;
   whee++;
   */  

   // preprocess column sizes for string columns.  the idea here is this:
   // - determining string lengths involves reaching into device memory to look at offsets, which is slow.
   // - contiguous_split() is typically used in situations with very large numbers of output columns, magnifying the slowness.
   // - so rather than reaching into device memory once per column (in column_buffer_size_functor), 
   //   we are doing it once per split.  For an example case of a table with 512 columns split 256 ways, that reduces
   //   our number of trips to/from the gpu from 128k -> 256

   // build a list of all the offset columns and their indices for all input string columns and put them on the gpu
   //
   // i'm using this pair structure instead of thrust::tuple because using tuple somehow causes the cudf::column_device_view
   // default constructor to get called (compiler error) when doing the assignment to device_offset_columns below
   thrust::host_vector<thrust::pair<thrust::pair<size_type, bool>, cudf::column_device_view>> offset_columns;
   offset_columns.reserve(t.num_columns());  // worst case
   size_type column_index = 0;
   std::for_each(t.begin(), t.end(), [&offset_columns, &column_index](cudf::column_view const& c){
      if(c.type().id() == STRING){
         // constructing device view from the offsets column only, because doing so for the entire
         // strings_column_view will result in memory allocation/hipMemcpy() calls, which would
         // defeat the whole purpose of this step.
         cudf::column_device_view cdv((strings_column_view(c)).offsets(), 0, 0);
         offset_columns.push_back(thrust::pair<thrust::pair<size_type, bool>, cudf::column_device_view>(
                  thrust::pair<size_type, bool>(column_index, c.nullable()), cdv));
      }
      column_index++;
   });   
   thrust::device_vector<thrust::pair<thrust::pair<size_type, bool>, cudf::column_device_view>> device_offset_columns = offset_columns;   

   // compute column sizes for all string columns
   thrust::device_vector<column_split_info> device_split_info(device_offset_columns.size());   
   auto *sizes_p = device_split_info.data().get();   
   thrust::for_each(rmm::exec_policy(stream)->on(stream), device_offset_columns.begin(), device_offset_columns.end(),
      [sizes_p] __device__ (auto column_info){
         size_type                  col_index = column_info.first.first;
         bool                       include_validity = column_info.first.second;
         cudf::column_device_view   col = column_info.second;
         size_type                  num_elements = col.size()-1;

         size_t align = split_align;

         auto num_chars = col.data<int32_t>()[num_elements] - col.data<int32_t>()[0];         
         sizes_p[col_index].data_size = round_up_safe_nothrow(static_cast<size_t>(num_chars), align);         
         // can't use cudf::bitmask_allocation_size_bytes() because it throws
         sizes_p[col_index].validity_size = include_validity ? bitmask_allocation_size_bytes_nothrow(num_elements, align) : 0;                  
         // can't use cudf::util::round_up_safe() because it throws
         sizes_p[col_index].offsets_size = round_up_safe_nothrow(col.size() * sizeof(size_type), align);
         sizes_p[col_index].chars_size = num_chars;
         sizes_p[col_index].chars_offset = col.data<int32_t>()[0];
      }
   );
   // copy sizes back from gpu. entries from non-string columns are uninitialized at this point.
   thrust::host_vector<column_split_info> split_info = device_split_info;
     
   // compute the rest of the column sizes (non-string columns, and total buffer size)
   size_t total_size = 0;
   column_index = 0;
   std::for_each(t.begin(), t.end(), [&total_size, &column_index, &split_info](cudf::column_view const& c){   
      total_size += cudf::experimental::type_dispatcher(c.type(), _column_buffer_size_functor{}, c, split_info[column_index]);
      column_index++;
   });

   /*   
   for(size_t idx=0; idx<column_sizes.size(); idx++){      
      printf("col %d,  (%d, %d, %d)\n", (int)idx, (int)thrust::get<0>(column_sizes[idx]), (int)thrust::get<1>(column_sizes[idx]), (int)thrust::get<2>(column_sizes[idx]));
      print_column(t.column(idx));
   }
   printf("Total size : %d\n", (int)total_size);
   */

   // allocate
   auto device_buf = std::make_unique<rmm::device_buffer>(total_size, stream, mr);
   char *buf = static_cast<char*>(device_buf->data());

   // copy (this would be cleaner with a std::transform, but there's an nvcc compiler issue in the way)   
   std::vector<column_view> out_cols;
   out_cols.reserve(t.num_columns());
   column_index = 0;   
   std::for_each(t.begin(), t.end(), [&out_cols, &buf, &column_index, &split_info](cudf::column_view const& c){
      cudf::experimental::type_dispatcher(c.type(), _column_copy_functor{}, c, split_info[column_index], buf, out_cols);
      column_index++;
   });   
   
   return contiguous_split_result{cudf::table_view{out_cols}, std::move(device_buf)};   
}

}; // anonymous namespace

std::vector<contiguous_split_result> _contiguous_split(cudf::table_view const& input,
                                                      std::vector<size_type> const& splits,
                                                      rmm::mr::device_memory_resource* mr,
                                                      hipStream_t stream)
{          
   auto subtables = cudf::experimental::split(input, splits);

/*
   for(int idx=0; idx<(int)subtables.size(); idx++){
      auto st = subtables[idx];
      auto col = st.column(0);      

      printf("table : %d\n", idx);
      
      auto offsets_col = col.child(0);
      auto chars_col = col.child(1);      
      printf("parent\nsize : %d\noffset : %d\n", col.size(), col.offset());
      printf("offsets\n");
      printf("   size : %d\n", offsets_col.size());
      printf("   offset : %d\n", offsets_col.offset());
      printf("   data (head) : %llx\n", (long long unsigned int) offsets_col.head());
      printf("   data (data) : %llx\n", (long long unsigned int) offsets_col.data<int32_t>());

      printf("chars\n");
      printf("   size : %d\n", chars_col.size());
      printf("   offset : %d\n", chars_col.offset());
      printf("   data (head) : %llx\n", (long long unsigned int) chars_col.head());
      printf("   data (data) : %llx\n", (long long unsigned int) chars_col.data<int32_t>());

      strings_column_view sv(col);
      auto offsets_col_s = sv.offsets();
      auto chars_col_s = sv.chars();      
      printf("offsets S\n");
      printf("   size : %d\n", offsets_col_s.size());
      printf("   offset : %d\n", offsets_col_s.offset());
      printf("   data (head) : %llx\n", (long long unsigned int) offsets_col_s.head());
      printf("   data (data) : %llx\n", (long long unsigned int) offsets_col_s.data<int32_t>());

      printf("chars S\n");
      printf("   size : %d\n", chars_col_s.size());
      printf("   offset : %d\n", chars_col_s.offset());
      printf("   data (head) : %llx\n", (long long unsigned int) chars_col_s.head());
      printf("   data (data) : %llx\n", (long long unsigned int) chars_col_s.data<int32_t>());
      
      printf("\n\n");
   }
   */   

   std::vector<contiguous_split_result> result;
   int idx = 0;
   std::transform(subtables.begin(), subtables.end(), std::back_inserter(result), [mr, stream, &idx](table_view const& t) { 
      idx++;
      return _alloc_and_copy(t, mr, stream);
   });

   return result;
}

}; // namespace detail

std::vector<contiguous_split_result> _contiguous_split(cudf::table_view const& input,
                                                      std::vector<size_type> const& splits,
                                                      rmm::mr::device_memory_resource* mr)
{    
   return cudf::experimental::detail::_contiguous_split(input, splits, mr, (hipStream_t)0);   
}

#endif

#if 0
namespace cudf {

namespace experimental {

namespace detail {

namespace {

using namespace::cudf;
using namespace::cudf::experimental;

template <typename S>
__device__ inline S round_up_safe_nothrow(S number_to_round, S modulus) {
    auto remainder = number_to_round % modulus;
    if (remainder == 0) { return number_to_round; }
    auto rounded_up = number_to_round - remainder + modulus;    
    return rounded_up;
}

// Computes required allocation size of a bitmask
__device__ std::size_t bitmask_allocation_size_bytes_nothrow(size_type number_of_bits,
                                          std::size_t padding_boundary) {  
  auto necessary_bytes =
      cudf::util::div_rounding_up_safe<size_type>(number_of_bits, CHAR_BIT);

  auto padded_bytes =
      padding_boundary * cudf::util::div_rounding_up_safe<size_type>(
                             necessary_bytes, padding_boundary);
  return padded_bytes;
}


/**
 * @brief Copies contents of `in` to `out`.  Copies validity if present
 * but does not compute null count.
 *  
 * @param in column_view to copy from
 * @param out mutable_column_view to copy to.
 */
template <size_type block_size, typename T, bool has_validity>
__launch_bounds__(block_size)
__global__
void copy_in_place_kernel( column_device_view const in,
                           mutable_column_device_view out)
{
   const size_type tid = threadIdx.x + blockIdx.x * block_size;
   const int warp_id = tid / cudf::experimental::detail::warp_size;
   const size_type warps_per_grid = gridDim.x * block_size / cudf::experimental::detail::warp_size;      

   // begin/end indices for the column data
   size_type begin = 0;
   size_type end = in.size();
   // warp indices.  since 1 warp == 32 threads == sizeof(bit_mask_t) * 8,
   // each warp will process one (32 bit) of the validity mask via
   // __ballot_sync()
   size_type warp_begin = cudf::word_index(begin);
   size_type warp_end = cudf::word_index(end-1);      

   // lane id within the current warp   
   const int lane_id = threadIdx.x % cudf::experimental::detail::warp_size;
   
   // current warp.
   size_type warp_cur = warp_begin + warp_id;   
   size_type index = tid;
   while(warp_cur <= warp_end){
      bool in_range = (index >= begin && index < end);
            
      bool valid = true;
      if(has_validity){
         valid = in_range && in.is_valid(index);
      }
      if(in_range){
         out.element<T>(index) = in.element<T>(index);
      }
      
      // update validity      
      if(has_validity){
         // the final validity mask for this warp 
         int warp_mask = __ballot_sync(0xFFFF'FFFF, valid && in_range);
         // only one guy in the warp needs to update the mask and count
         if(lane_id == 0){            
            out.set_mask_word(warp_cur, warp_mask);            
         }
      }            

      // next grid
      warp_cur += warps_per_grid;
      index += block_size * gridDim.x;
   }      
}

template <size_type block_size, bool has_validity>
__launch_bounds__(block_size)
__global__
void copy_in_place_strings_kernel(size_type                        num_rows,
                                  size_type const* __restrict__    offsets_in,
                                  size_type* __restrict__          offsets_out,
                                  size_type                        validity_in_offset,
                                  bitmask_type const* __restrict__ validity_in,
                                  bitmask_type* __restrict__       validity_out,

                                  size_type                        offset_shift,

                                  size_type                        num_chars,
                                  char const* __restrict__         chars_in,
                                  char* __restrict__               chars_out)
{   
   const size_type tid = threadIdx.x + blockIdx.x * block_size;
   const int warp_id = tid / cudf::experimental::detail::warp_size;
   const size_type warps_per_grid = gridDim.x * block_size / cudf::experimental::detail::warp_size;   
   
   // how many warps we'll be processing. with strings, the chars and offsets
   // lengths may be different.  so we'll just march the worst case.
   size_type warp_begin = cudf::word_index(0);
   size_type warp_end = cudf::word_index(std::max(num_chars, num_rows+1)-1);

   // end indices for chars   
   size_type chars_end = num_chars;
   // end indices for offsets   
   size_type offsets_end = num_rows+1;
   // end indices for validity and the last warp that actually should
   // be updated
   size_type validity_end = num_rows;
   size_type validity_warp_end = cudf::word_index(num_rows-1);  

   // lane id within the current warp   
   const int lane_id = threadIdx.x % cudf::experimental::detail::warp_size;

   size_type warp_cur = warp_begin + warp_id;
   size_type index = tid;
   while(warp_cur <= warp_end){      
      if(index < chars_end){
         chars_out[index] = chars_in[index];
      }
      
      if(index < offsets_end){
         // each output column starts at a new base pointer. so we have to
         // shift every offset down by the point (in chars) at which it was split.
         offsets_out[index] = offsets_in[index] - offset_shift;
      }

      // if we're still in range of validity at all
      if(has_validity && warp_cur <= validity_warp_end){
         bool valid = (index < validity_end) && bit_is_set(validity_in, validity_in_offset + index);
               
         // the final validity mask for this warp 
         int warp_mask = __ballot_sync(0xFFFF'FFFF, valid);
         // only one guy in the warp needs to update the mask and count
         if(lane_id == 0){                        
            validity_out[warp_cur] = warp_mask;
         }
      }            

      // next grid
      warp_cur += warps_per_grid;
      index += block_size * gridDim.x;
   }    
}

// align all column size allocations to this boundary so that all output column buffers
// start at that alignment.
static constexpr size_t split_align = 64;

struct column_split_info {
   size_type   data_size;     // size of the data
   size_type   validity_size; // validity vector size
   
   size_type   offsets_size;  // (strings only) size of offset column
   size_type   chars_size;    // (strings only) # of chars in the column
   size_type   chars_offset;  // (strings only) offset from head of chars data
};

/**
 * @brief Functor called by the `type_dispatcher` to incrementally compute total
 * memory buffer size needed to allocate a contiguous copy of all columns within
 * a source table. 
 */
struct column_buffer_size_functor {
   template <typename T, std::enable_if_t<not is_fixed_width<T>()>* = nullptr>
   size_t operator()(column_view const& c, column_split_info &split_info)
   {
      // this has already been precomputed in an earlier step      
      return split_info.data_size + split_info.validity_size + split_info.offsets_size;
   }

   template <typename T, std::enable_if_t<is_fixed_width<T>()>* = nullptr>
   size_t operator()(column_view const& c, column_split_info &split_info)
   {      
      split_info.data_size = cudf::util::round_up_safe(c.size() * sizeof(T), split_align);  
      split_info.validity_size = (c.nullable() ? cudf::bitmask_allocation_size_bytes(c.size(), split_align) : 0);
      return split_info.data_size + split_info.validity_size;
   }
};

/**
 * @brief Functor called by the `type_dispatcher` to copy a column into a contiguous
 * buffer of output memory. 
 * 
 * Used for copying each column in a source table into one contiguous buffer of memory.
 */
struct column_copy_functor {
   template <typename T, std::enable_if_t<not is_fixed_width<T>()>* = nullptr>
   void operator()(column_view const& in, column_split_info const& split_info, char*& dst, std::vector<column_view>& out_cols)
   {            
      // outgoing pointers
      char* chars_buf = dst;
      bitmask_type* validity_buf = split_info.validity_size == 0 ? nullptr : reinterpret_cast<bitmask_type*>(dst + split_info.data_size);
      size_type* offsets_buf = reinterpret_cast<size_type*>(dst + split_info.data_size + split_info.validity_size);

      // increment working buffer
      dst += (split_info.data_size + split_info.validity_size + split_info.offsets_size);

      // offsets column
      strings_column_view strings_c(in);
      column_view in_offsets = strings_c.offsets();
      // get the chars column directly instead of calling .chars(), since .chars() will end up
      // doing gpu work we specifically want to avoid.
      column_view in_chars = in.child(strings_column_view::chars_column_index);      
      
      // 1 combined kernel call that copies chars, offsets and validity in one pass
      cudf::size_type num_els = cudf::util::round_up_safe(std::max(split_info.chars_size, in_offsets.size() + 1)/*strings_c.offsets().size()*/, cudf::experimental::detail::warp_size);
      constexpr int block_size = 256;
      cudf::experimental::detail::grid_1d grid{num_els, block_size, 1};            
      if(in.nullable()){
         copy_in_place_strings_kernel<block_size, true><<<grid.num_blocks, block_size, 0, 0>>>(
                           in.size(),                                            // num_rows
                           in_offsets.data<size_type>(),                         // offsets_in
                           offsets_buf,                                          // offsets_out
                           in.offset(),                                          // validity_in_offset
                           in.null_mask(),                                       // validity_in
                           validity_buf,                                         // validity_out

                           split_info.chars_offset,                              // offset_shift

                           split_info.chars_size,                                // num_chars
                           in_chars.head<char>() + split_info.chars_offset,      // chars_in
                           chars_buf);                                                      
      } else {                                       
         copy_in_place_strings_kernel<block_size, false><<<grid.num_blocks, block_size, 0, 0>>>(
                           in.size(),                                            // num_rows
                           in_offsets.data<size_type>(),                         // offsets_in
                           offsets_buf,                                          // offsets_out
                           0,                                                    // validity_in_offset
                           nullptr,                                              // validity_in
                           nullptr,                                              // validity_out

                           split_info.chars_offset,                              // offset_shift

                           split_info.chars_size,                                // num_chars
                           in_chars.head<char>() + split_info.chars_offset,      // chars_in
                           chars_buf);                                                      
      }      

      // output child columns      
      column_view out_offsets{strings_c.offsets().type(), strings_c.offsets().size(), offsets_buf};
      column_view out_chars{in_chars.type(), static_cast<size_type>(split_info.chars_size), chars_buf};      

      // result
      out_cols.push_back(column_view(in.type(), in.size(), nullptr,
                                     validity_buf, UNKNOWN_NULL_COUNT, 0,
                                     { out_offsets, out_chars }));                                     

      /*
      strings_column_view strings_c(in);

      // outgoing pointers
      char* chars_buf = dst;
      bitmask_type* validity_buf = split_info.validity_size == 0 ? nullptr : reinterpret_cast<bitmask_type*>(dst + split_info.data_size);
      size_type* offsets_buf = reinterpret_cast<size_type*>(dst + split_info.data_size + split_info.validity_size);

      // increment working buffer
      dst += (split_info.data_size + split_info.validity_size + split_info.offsets_size);
                                     
      // copy offsets and validity
      column_view offsets_col = strings_c.offsets();
      mutable_column_view temp_offsets_and_validity{
                              offsets_col.type(), offsets_col.size(), offsets_buf,
                              validity_buf, validity_buf == nullptr ? UNKNOWN_NULL_COUNT : 0,
                              0 };
      {         
         // contruct a column which wraps the validity vector and the offsets from the child column. 
         // this is weird but it removes an extra kernel call. however, since the length of the offsets column
         // is always 1 greater than the # of strings, the validity vector will be short by 1. the kernel will have to
         // compensate for that. 
         CUDF_EXPECTS(in.size() == offsets_col.size()-1, "Expected offsets column to be the same size as parent");
         CUDF_EXPECTS(in.offset() == offsets_col.offset(), "Expected offsets column offset to be the same as parent");
         CUDF_EXPECTS(offsets_col.type() == cudf::data_type(INT32), "Expected offsets column type to be int32");
         column_view in_offsets_and_validity{
                                 offsets_col.type(), offsets_col.size(), offsets_col.head<int32_t>(),
                                 in.null_mask(), in.null_mask() == nullptr ? UNKNOWN_NULL_COUNT : 0,
                                 in.offset()};
         
         cudf::size_type num_els = cudf::util::round_up_safe(strings_c.offsets().size(), cudf::experimental::detail::warp_size);
         constexpr int block_size = 256;
         cudf::experimental::detail::grid_1d grid{num_els, block_size, 1};
         if(in.nullable()){
            copy_in_place_kernel<block_size, size_type, true><<<grid.num_blocks, block_size, 0, 0>>>(
                              *column_device_view::create(in_offsets_and_validity), 
                              in.size(),  // validity vector length
                              *mutable_column_device_view::create(temp_offsets_and_validity), split_info.chars_offset);
         } else {
            copy_in_place_kernel<block_size, size_type, false><<<grid.num_blocks, block_size, 0, 0>>>(
                              *column_device_view::create(in_offsets_and_validity),
                              in.size(),  // validity vector length
                              *mutable_column_device_view::create(temp_offsets_and_validity), split_info.chars_offset);
         }
      }

      // get the chars column directly instead of calling .chars(), since .chars() will end up
      // doing gpu work we specifically want to avoid.
      column_view chars_col = in.child(strings_column_view::chars_column_index);

      // copy chars
      mutable_column_view out_chars{chars_col.type(), static_cast<size_type>(split_info.chars_size), chars_buf};      
      {         
         CUDF_EXPECTS(!chars_col.nullable(), "Expected input chars column to not be nullable");
         CUDF_EXPECTS(chars_col.offset() == 0, "Expected input chars column to have an offset of 0");
         column_view in_chars{ chars_col.type(), static_cast<size_type>(split_info.chars_size), chars_col.data<char>() + split_info.chars_offset };
                                 
         cudf::size_type num_els = cudf::util::round_up_safe(static_cast<size_type>(split_info.chars_size), cudf::experimental::detail::warp_size);
         constexpr int block_size = 256;
         cudf::experimental::detail::grid_1d grid{num_els, block_size, 1};         
         copy_in_place_kernel<block_size, char, false><<<grid.num_blocks, block_size, 0, 0>>>(
                           *column_device_view::create(in_chars),
                           split_info.chars_size,
                           *mutable_column_device_view::create(out_chars), 0);
      }
      */
   }

   template <typename T, std::enable_if_t<is_fixed_width<T>()>* = nullptr>
   void operator()(column_view const& in, column_split_info const& split_info, char*& dst, std::vector<column_view>& out_cols)
   {     
      // outgoing pointers
      char* data = dst;
      bitmask_type* validity = split_info.validity_size == 0 ? nullptr : reinterpret_cast<bitmask_type*>(dst + split_info.data_size);

      // increment working buffer
      dst += (split_info.data_size + split_info.validity_size);

      // custom copy kernel (which should probably just be an in-place copy() function in cudf.
      cudf::size_type num_els = cudf::util::round_up_safe(in.size(), cudf::experimental::detail::warp_size);
      constexpr int block_size = 256;
      cudf::experimental::detail::grid_1d grid{num_els, block_size, 1};
      
      // so there's a significant performance issue that comes up. our incoming column_view objects
      // are the result of a slice.  because of this, they have an UNKNOWN_NULL_COUNT.  because of that,
      // calling column_device_view::create() will cause a recompute of the count, which ends up being
      // extremely slow because a.) the typical use case here will involve huge numbers of calls and
      // b.) the count recompute involves tons of device allocs and memcopies.
      //
      // so to get around this, I am manually constructing a fake-ish view here where the null
      // count is arbitrarily bashed to 0.            
      //            
      // Remove this hack once rapidsai/cudf#3600 is fixed.
      column_view   in_wrapped{in.type(), in.size(), in.head<T>(), 
                               in.null_mask(), in.null_mask() == nullptr ? UNKNOWN_NULL_COUNT : 0,
                               in.offset() };
      mutable_column_view  mcv{in.type(), in.size(), data, 
                               validity, validity == nullptr ? UNKNOWN_NULL_COUNT : 0 };      
      if(in.nullable()){               
         copy_in_place_kernel<block_size, T, true><<<grid.num_blocks, block_size, 0, 0>>>(
                           *column_device_view::create(in_wrapped),                            
                           *mutable_column_device_view::create(mcv));         
      } else {
         copy_in_place_kernel<block_size, T, false><<<grid.num_blocks, block_size, 0, 0>>>(
                           *column_device_view::create(in_wrapped),                            
                           *mutable_column_device_view::create(mcv));
      }
      mcv.set_null_count(cudf::UNKNOWN_NULL_COUNT);                 

      out_cols.push_back(mcv);
   }
};

/**
 * @brief Creates a contiguous_split_result object which contains a deep-copy of the input
 * table_view into a single contiguous block of memory. 
 * 
 * The table_view contained within the contiguous_split_result will pass an expect_tables_equal()
 * call with the input table.  The memory referenced by the table_view and its internal column_views
 * is entirely contained in single block of memory.
 */
contiguous_split_result alloc_and_copy(cudf::table_view const& t, thrust::device_vector<column_split_info>& device_split_info, rmm::mr::device_memory_resource* mr, hipStream_t stream)      
{            
   // preprocess column sizes for string columns.  the idea here is this:
   // - determining string lengths involves reaching into device memory to look at offsets, which is slow.
   // - contiguous_split() is typically used in situations with very large numbers of output columns, exaggerating
   //   the problem.
   // - so rather than reaching into device memory once per column (in column_buffer_size_functor), 
   //   we are doing it once per split (for all string columns in the split).  For an example case of a table with 
   //   512 columns split 256 ways, that reduces our number of trips to/from the gpu from 128k -> 256
   
   // build a list of all the offset columns and their indices for all input string columns and put them on the gpu
   //
   // i'm using this pair structure instead of thrust::tuple because using tuple somehow causes the cudf::column_device_view
   // default constructor to get called (compiler error) when doing the assignment to device_offset_columns below
   thrust::host_vector<thrust::pair<thrust::pair<size_type, bool>, cudf::column_device_view>> offset_columns;
   offset_columns.reserve(t.num_columns());  // worst case
   size_type column_index = 0;
   std::for_each(t.begin(), t.end(), [&offset_columns, &column_index](cudf::column_view const& c){
      if(c.type().id() == STRING){
         // constructing device view from the offsets column only, because doing so for the entire
         // strings_column_view will result in memory allocation/hipMemcpy() calls, which would
         // defeat the whole purpose of this step.
         cudf::column_device_view cdv((strings_column_view(c)).offsets(), 0, 0);
         offset_columns.push_back(thrust::pair<thrust::pair<size_type, bool>, cudf::column_device_view>(
                  thrust::pair<size_type, bool>(column_index, c.nullable()), cdv));
      }
      column_index++;
   });   
   thrust::device_vector<thrust::pair<thrust::pair<size_type, bool>, cudf::column_device_view>> device_offset_columns = offset_columns;   
     
   // compute column sizes for all string columns   
   auto *sizes_p = device_split_info.data().get();   
   thrust::for_each(rmm::exec_policy(stream)->on(stream), device_offset_columns.begin(), device_offset_columns.end(),
      [sizes_p] __device__ (auto column_info){
         size_type                  col_index = column_info.first.first;
         bool                       include_validity = column_info.first.second;
         cudf::column_device_view   col = column_info.second;
         size_type                  num_elements = col.size()-1;

         size_t align = split_align;

         auto num_chars = col.data<int32_t>()[num_elements] - col.data<int32_t>()[0];
         sizes_p[col_index].data_size = round_up_safe_nothrow(static_cast<size_t>(num_chars), align);         
         // can't use cudf::bitmask_allocation_size_bytes() because it throws
         sizes_p[col_index].validity_size = include_validity ? bitmask_allocation_size_bytes_nothrow(num_elements, align) : 0;
         // can't use cudf::util::round_up_safe() because it throws
         sizes_p[col_index].offsets_size = round_up_safe_nothrow(col.size() * sizeof(size_type), align);
         sizes_p[col_index].chars_size = num_chars;
         sizes_p[col_index].chars_offset = col.data<int32_t>()[0];
      }
   );
   
   // copy sizes back from gpu. entries from non-string columns are uninitialized at this point.
   thrust::host_vector<column_split_info> split_info = device_split_info;     
     
   // compute the rest of the column sizes (non-string columns, and total buffer size)
   size_t total_size = 0;
   column_index = 0;
   std::for_each(t.begin(), t.end(), [&total_size, &column_index, &split_info](cudf::column_view const& c){   
      total_size += cudf::experimental::type_dispatcher(c.type(), column_buffer_size_functor{}, c, split_info[column_index]);
      column_index++;
   });

   // allocate
   auto device_buf = std::make_unique<rmm::device_buffer>(total_size, stream, mr);
   char *buf = static_cast<char*>(device_buf->data());

   // copy (this would be cleaner with a std::transform, but there's an nvcc compiler issue in the way)   
   std::vector<column_view> out_cols;
   out_cols.reserve(t.num_columns());
   column_index = 0;   
   std::for_each(t.begin(), t.end(), [&out_cols, &buf, &column_index, &split_info](cudf::column_view const& c){
      cudf::experimental::type_dispatcher(c.type(), column_copy_functor{}, c, split_info[column_index], buf, out_cols);
      column_index++;
   });   
   
   return contiguous_split_result{cudf::table_view{out_cols}, std::move(device_buf)};   
}

}; // anonymous namespace

std::vector<contiguous_split_result> _contiguous_split(cudf::table_view const& input,
                                                      std::vector<size_type> const& splits,
                                                      rmm::mr::device_memory_resource* mr,
                                                      hipStream_t stream)
{   
   auto subtables = cudf::experimental::split(input, splits);

   // optimization : for large #'s of splits this allocation can dominate total time
   //                spent if done inside alloc_and_copy().  so we'll allocate it once
   //                and reuse it.
   // 
   //                benchmark:        1 GB data, 10 columns, 256 splits.
   //                no optimization:  106 ms (8 GB/s)
   //                optimization:     20 ms (48 GB/s)
   thrust::device_vector<column_split_info> device_split_info(input.num_columns());

   std::vector<contiguous_split_result> result;
   std::transform(subtables.begin(), subtables.end(), std::back_inserter(result), [mr, stream, &device_split_info](table_view const& t) { 
      return alloc_and_copy(t, device_split_info, mr, stream);
   });

   return result;
}

}; // namespace detail

std::vector<contiguous_split_result> _contiguous_split(cudf::table_view const& input,
                                                      std::vector<size_type> const& splits,
                                                      rmm::mr::device_memory_resource* mr)
{    
   return cudf::experimental::detail::_contiguous_split(input, splits, mr, (hipStream_t)0);   
}

}; // namespace experimental

}; // namespace cudf

#endif

using namespace cudf;
using namespace cudf::experimental;

namespace {
std::vector<cudf::size_type> splits_to_indices(std::vector<cudf::size_type> splits, cudf::size_type size){
    std::vector<cudf::size_type> indices{0};

    std::for_each(splits.begin(), splits.end(),
            [&indices](auto split) {
                indices.push_back(split); // This for end
                indices.push_back(split); // This for the start
            });

    if (splits.back() != size) {
        indices.push_back(size); // This to include rest of the elements
    } else {
        indices.pop_back(); // Not required as it is extra 
    }

    return indices;
}

void verify_split_results( cudf::experimental::table const& src_table, 
                           std::vector<contiguous_split_result> const &dst_tables,
                           std::vector<size_type> const& splits,
                           int verbosity = 0)
{     
   table_view src_v(src_table.view());
   
   int col_count = 0;
   for(size_t c_idx = 0; c_idx<(size_t)src_v.num_columns(); c_idx++){
      // grab this column from each subtable
      auto src_subcols = cudf::experimental::split(src_v.column(c_idx), splits);

      for(size_t t_idx=0; t_idx<src_subcols.size(); t_idx++){
         cudf::test::expect_columns_equal(src_subcols[t_idx], dst_tables[t_idx].table.column(c_idx), true);
         
         if(verbosity > 0 && (col_count % verbosity == 0)){
            printf("----------------------------\n");            
            print_column(src_subcols[t_idx], false, 20);
            print_column(dst_tables[t_idx].table.column(c_idx), false, 20);
            printf("----------------------------\n");
         }
         col_count++;
      }
   }      
}

float frand()
{
   return (static_cast<float>(rand()) / static_cast<float>(RAND_MAX)) * 65535.0f;
}

int rand_range(int r)
{
   return static_cast<int>((static_cast<float>(rand()) / static_cast<float>(RAND_MAX)) * (float)(r-1));
}

template<typename T>
void single_split_test_common(std::vector<T>& src_cols, 
                              int64_t num_cols, 
                              int64_t num_rows, 
                              int64_t num_splits,
                              int verbosity)
{
   scope_timer_manual null_count_gen("null count gen");
   null_count_gen.start();

   std::vector<std::unique_ptr<column>> columns((size_t)num_cols);
   std::transform(src_cols.begin(), src_cols.end(), columns.begin(), [](T& in){   
      auto ret = in.release();
      ret->null_count();
      return ret;
   });
   null_count_gen.end();

   cudf::experimental::table src_table(std::move(columns));   
   // print_table(src_table);
   printf("# columns : %d\n", (int)num_cols);
   
   // generate splits 
   int split_stride = num_rows / num_splits;
   std::vector<size_type> splits;  
   scope_timer_manual split_gen("split gen");
   split_gen.start();
   for(size_t idx=0; idx<(size_t)num_rows; idx+=split_stride){      
      splits.push_back(min((int)(idx + split_stride), (int)num_rows));
   }
   split_gen.end();
     
   // do the split
   scope_timer_manual split_time("contiguous_split total");
   split_time.start();   
   auto dst_tables = cudf::experimental::contiguous_split(src_table.view(), splits, rmm::mr::get_default_resource());
   hipDeviceSynchronize();
   split_time.end();

   scope_timer_manual verify_time("verify_split_results");
   verify_time.start();
   verify_split_results(src_table, dst_tables, splits, verbosity);
   verify_time.end();

   scope_timer_manual free_time("free buffers");
   free_time.start();   
   for(size_t idx=0; idx<dst_tables.size(); idx++){
      rmm::device_buffer *buf = dst_tables[idx].all_data.release();
      delete buf;   
   }
   hipDeviceSynchronize();
   free_time.end();            
}

void single_split_test( int64_t total_desired_bytes, 
                        int64_t num_cols,                     
                        int64_t num_rows,
                        int64_t num_splits,
                        bool include_validity)
{
   printf("total data size : %.2f GB\n", (float)total_desired_bytes / (float)(1024 * 1024 * 1024));
   
   srand(31337);

   // generate input columns and table      
   scope_timer_manual src_table_gen("src table gen");   
   src_table_gen.start();

   auto valids = cudf::test::make_counting_transform_iterator(0, [](auto i) { return i%2 == 0 ? true : false; });
   std::vector<cudf::test::fixed_width_column_wrapper<int>> src_cols(num_cols);
   for(int idx=0; idx<num_cols; idx++){
      auto rand_elements = cudf::test::make_counting_transform_iterator(0, [](int i){return rand();});
      if(include_validity){
         src_cols[idx] = cudf::test::fixed_width_column_wrapper<int>(rand_elements, rand_elements + num_rows, valids);
      } else {
         src_cols[idx] = cudf::test::fixed_width_column_wrapper<int>(rand_elements, rand_elements + num_rows);
      }
   }      
   src_table_gen.end();

   single_split_test_common(src_cols, num_cols, num_rows, num_splits, 1000);   
}

void single_split_test_string( int64_t total_desired_bytes,
                               int64_t num_cols,                        
                               int64_t num_splits,
                               bool include_validity)
{     
   printf("total data size : %.2f GB\n", (float)total_desired_bytes / (float)(1024 * 1024 * 1024));
   
   srand(31337);

   // generate input columns and table      
   scope_timer_manual src_table_gen("src table gen");   
   src_table_gen.start();
   
   // const int64_t string_len[8] = { 8, 4, 5, 7, 2, 3, 8, 6 };
   const int64_t avg_string_len = 6;   // eh. don't really need to hit the # of bytes exactly. just ballpark
   std::vector<const char*> h_strings{ "aaaaaaaa", "b", "ccccc", "ddddddd", "ee", "fff", "gggggggg", "hhhhhh" };   
   
   int64_t col_len_bytes = total_desired_bytes / num_cols;
   int64_t num_rows = col_len_bytes / avg_string_len;      
      
   // generate table
   auto valids = cudf::test::make_counting_transform_iterator(0, [](auto i) { return i%2 == 0 ? true : false; });   
   std::vector<cudf::test::strings_column_wrapper> src_cols;
   {
      std::vector<const char*> one_col(num_rows);
      for(int64_t idx=0; idx<num_cols; idx++){
         // fill in a random set of strings
         for(int64_t s_idx=0; s_idx<num_rows; s_idx++){
            one_col[s_idx] = h_strings[rand_range(h_strings.size())];         
         }
         if(include_validity){
            src_cols.push_back(cudf::test::strings_column_wrapper(one_col.begin(), one_col.end(), valids));
         } else {
            src_cols.push_back(cudf::test::strings_column_wrapper(one_col.begin(), one_col.end()));
         }
      }
   }
   src_table_gen.end();

   single_split_test_common(src_cols, num_cols, num_rows, num_splits, 1000);
}

};

void large_split_tests()
{      
   // single_split_test does ints and floats only
   int el_size = 4;
      
   /*
   {
      // Tesla T4, 16 GB (all times in milliseconds)
      // total data size : 2.00 GB
      // src table gen : 8442.80 ms
      // # columns : 512
      // split gen : 0.00 ms
      // # splits : 256
      //    alloc time : 77.27 (256 allocs)     <------
      //    copy time : 436.92 (131072 copies)  <------
      // contiguous_split total : 524.31 ms     <------
      // verify_split_results : 6763.76 ms
      // free buffers : 0.18 ms                 <------
 
      // pick some numbers
      int64_t total_desired_bytes = (int64_t)2 * (1024 * 1024 * 1024);
      int64_t num_cols = 512;      
      int64_t num_rows = total_desired_bytes / (num_cols * el_size);
      int64_t num_splits = num_cols / 2;
      single_split_test(total_desired_bytes, num_cols, num_rows, num_splits, false);
   }
   */  
   
   /*
   {
      // Tesla T4, 16 GB (all times in milliseconds)
      // total data size : 2.00 GB
      // src table gen : 9383.00 ms
      // # columns : 512
      // split gen : 0.00 ms
      // # splits : 256
      //    alloc time : 43.93 (256 allocs)     <------
      //    copy time : 413.77 (131072 copies)  <------
      // contiguous_split total : 469.21 ms     <------
      // verify_split_results : 11387.72 ms
      // free buffers : 0.20 ms                 <------
 
      // pick some numbers
      int64_t total_desired_bytes = (int64_t)2 * (1024 * 1024 * 1024);
      int64_t num_cols = 512;      
      int64_t num_rows = total_desired_bytes / (num_cols * el_size);
      int64_t num_splits = num_cols / 2;
      single_split_test(total_desired_bytes, num_cols, num_rows, num_splits, true);
   }
   */  

   /*      
   {
      // Tesla T4, 16 GB (all times in milliseconds)
      // total data size : 4.00 GB
      // src table gen : 16917.02 ms
      // # columns : 512
      // split gen : 0.00 ms
      // # splits : 256
      //    alloc time : 79.27 (256 allocs)     <------
      //    copy time : 454.59 (131072 copies)  <------
      // contiguous_split total : 541.54 ms     <------
      // verify_split_results : 6777.47 ms
      // free buffers : 0.18 ms                 <------
 
      // pick some numbers
      int64_t total_desired_bytes = (int64_t)4 * (1024 * 1024 * 1024);
      int64_t num_cols = 512;      
      int64_t num_rows = total_desired_bytes / (num_cols * el_size);
      int64_t num_splits = num_cols / 2;
      single_split_test(total_desired_bytes, num_cols, num_rows, num_splits, false);
   } 
   */        

   /* 
   {
      // Tesla T4, 16 GB (all times in milliseconds)
      // total data size : 4.00 GB
      // src table gen : 18649.68 ms
      // # columns : 512
      // split gen : 0.00 ms
      // # splits : 256
      //    alloc time : 47.73 (256 allocs)     <------
      //    copy time : 446.58 (131072 copies)  <------
      // contiguous_split total : 503.26 ms     <------
      // verify_split_results : 11802.98 ms
      // free buffers : 0.26 ms                 <------
 
      // pick some numbers
      int64_t total_desired_bytes = (int64_t)4 * (1024 * 1024 * 1024);
      int64_t num_cols = 512;      
      int64_t num_rows = total_desired_bytes / (num_cols * el_size);
      int64_t num_splits = num_cols / 2;
      single_split_test(total_desired_bytes, num_cols, num_rows, num_splits, true);
   }   
   */
   
   /*
   {
      // Tesla T4, 16 GB
      // total data size : 6.00 GB
      // src table gen : 25230.81 ms
      // # columns : 512
      // split gen : 0.00 ms
      // # splits : 256
      //    alloc time : 48.01 (256 allocs)     <------
      //    copy time : 416.30 (131072 copies)  <------
      // contiguous_split total : 471.48 ms     <------
      // verify_split_results : 53921.47 ms
      // free buffers : 0.20 ms                 <------

      // pick some numbers
      int64_t total_desired_bytes = (int64_t)1 * (1024 * 1024 * 1024);
      int64_t num_cols = 512;      
      int64_t num_rows = total_desired_bytes / (num_cols * el_size);
      int64_t num_splits = num_cols / 2;
      single_split_test(total_desired_bytes, num_cols, num_rows, num_splits, true);
   }   
   */ 
      
   /*
   {
      // Tesla T4, 16 GB
      // total data size : 6.00 GB
      // src table gen : 27897.44 ms
      // # columns : 512
      // split gen : 0.00 ms
      // # splits : 256
      //    alloc time : 61.25 (256 allocs)     <------
      //    copy time : 447.05 (131072 copies)  <------
      // contiguous_split total : 517.05 ms     <------
      // verify_split_results : 13794.44 ms
      // free buffers : 0.20 ms                 <------
 
      // pick some numbers
      int64_t total_desired_bytes = (int64_t)6 * (1024 * 1024 * 1024);
      int64_t num_cols = 512;      
      int64_t num_rows = total_desired_bytes / (num_cols * el_size);
      int64_t num_splits = num_cols / 2;
      single_split_test(total_desired_bytes, num_cols, num_rows, num_splits, true);
   }
   */
   
   /*
   {
      // Tesla T4, 16 GB
      // total data size : 6.00 GB
      // src table gen : 28402.29 ms
      // # columns : 10
      // split gen : 0.01 ms
      // # splits : 257
      //    alloc time : 45.74 (257 allocs)     <------
      //    copy time : 70.60 (2570 copies)     <------
      // contiguous_split total : 116.76 ms     <------
      // verify_split_results : 1962.77 ms
      // free buffers : 0.24 ms                 <------
 
      // pick some numbers
      int64_t total_desired_bytes = (int64_t)6 * (1024 * 1024 * 1024);
      int64_t num_cols = 10;
      int64_t num_rows = total_desired_bytes / (num_cols * el_size);
      int64_t num_splits = 256;
      single_split_test(total_desired_bytes, num_cols, num_rows, num_splits, false);
   }
   */

      /*
    {
      // Tesla T4, 16 GB
      // total data size : 6.00 GB
      // src table gen : 30930.70 ms
      // # columns : 10
      // split gen : 0.00 ms
      // # splits : 257
      //    alloc time : 42.77 (257 allocs)     <------
      //    copy time : 72.51 (2570 copies)     <------
      // contiguous_split total : 115.61 ms     <------
      // verify_split_results : 2088.58 ms
      // free buffers : 0.25 ms                 <------
 
      // pick some numbers
      int64_t total_desired_bytes = (int64_t)2 * (1024 * 1024 * 1024);
      int64_t num_cols = 10;
      int64_t num_rows = total_desired_bytes / (num_cols * el_size);
      int64_t num_splits = 256;
      single_split_test(total_desired_bytes, num_cols, num_rows, num_splits, true);
   }   
   */

   {
      // Tesla T4, 16 GB
      // total data size : 6.00 GB
      // src table gen : 25230.81 ms
      // # columns : 512
      // split gen : 0.00 ms
      // # splits : 256
      //    alloc time : 48.01 (256 allocs)     <------
      //    copy time : 416.30 (131072 copies)  <------
      // contiguous_split total : 471.48 ms     <------
      // verify_split_results : 53921.47 ms
      // free buffers : 0.20 ms                 <------

      // pick some numbers
      int64_t total_desired_bytes = (int64_t)4 * (1024 * 1024 * 1024);      
      // int64_t total_desired_bytes = (int64_t)(512 * 1024 * 1024);
      int64_t num_cols = 512;            
      int64_t num_splits = num_cols / 2;
      single_split_test_string(total_desired_bytes, num_cols, num_splits, true);
   }   
}

inline std::vector<cudf::experimental::table> create_expected_string_tables(std::vector<std::string> const strings[2], std::vector<cudf::size_type> const& indices, bool nullable) {

    std::vector<cudf::experimental::table> result = {};

    for(unsigned long index = 0; index < indices.size(); index+=2) {
        std::vector<std::unique_ptr<cudf::column>> cols = {};
        
        for(int idx=0; idx<2; idx++){     
            if(not nullable) {
                cudf::test::strings_column_wrapper wrap(strings[idx].begin()+indices[index], strings[idx].begin()+indices[index+1]);                
                cols.push_back(wrap.release());
            } else {
                auto valids = cudf::test::make_counting_transform_iterator(indices[index], [](auto i) { return i % 2 == 0 ? true : false; });
                cudf::test::strings_column_wrapper wrap(strings[idx].begin()+indices[index], strings[idx].begin()+indices[index+1], valids);
                cols.push_back(wrap.release());
            }
        }

        result.push_back(cudf::experimental::table(std::move(cols)));
    }

    return result;
}

std::vector<cudf::experimental::table> create_expected_string_tables_for_splits(std::vector<std::string> const strings[2], std::vector<cudf::size_type> const& splits, bool nullable){    
    std::vector<cudf::size_type> indices = splits_to_indices(splits, strings[0].size());    
    return create_expected_string_tables(strings, indices, nullable);
}

std::unique_ptr<column> make_strings(std::vector<const char*> _strings)
{   
   cudf::test::strings_column_wrapper strings( _strings.begin(), _strings.end(),
        thrust::make_transform_iterator( _strings.begin(), [] (auto str) { return str!=nullptr; }));
      
   return strings.release();
}

void split_test()
{   
   /*
   std::vector<std::unique_ptr<column>> columns;
   int c0d[] = { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9 };
   wrapper<int> c0(c0d, c0d + 10);
   columns.push_back(c0.release());

   short c1d[] = { 20, 21, 22, 23, 24, 25, 26, 27, 28, 29 };
   wrapper<short> c1(c1d, c1d + 10);
   columns.push_back(c1.release());

   double c2d[] = { 30, 31, 32, 33, 34, 35, 36, 37, 38, 39 };
   wrapper<double> c2(c2d, c2d + 10);
   columns.push_back(c2.release());
   
   cudf::experimental::table t(std::move(columns));
   print_table(t.view());

   std::vector<size_type> splits { 5, 10 };

   auto out = contiguous_split(t.view(), splits);
   
   for(size_t idx=0; idx<out.size(); idx++){
      print_table(out[idx].table);
   }
   
   int whee = 10;
   whee++;
   */
   
   /*
   std::vector<std::unique_ptr<column>> columns;
   int c0d[] = { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9 };
   bool c0v[] ={ 1, 1, 1, 1, 0, 0, 1, 1, 1, 1 };
   wrapper<int> c0(c0d, c0d + 10, c0v);
   columns.push_back(c0.release());

   short c1d[] = { 20, 21, 22, 23, 24, 25, 26, 27, 28, 29 };
   bool c1v[] ={ 1, 1, 1, 0, 1, 1, 0, 1, 1, 1 };
   wrapper<short> c1(c1d, c1d + 10, c1v);
   columns.push_back(c1.release());

   double c2d[] = { 30, 31, 32, 33, 34, 35, 36, 37, 38, 39 };
   bool c2v[] ={ 1, 1, 0, 1, 1, 1, 1, 0, 1, 1 };
   wrapper<double> c2(c2d, c2d + 10, c2v);
   columns.push_back(c2.release());
   
   cudf::experimental::table t(std::move(columns));
   print_table(t.view());

   std::vector<size_type> splits { 5, 10 };

   auto out = cudf::experimental::contiguous_split(t.view(), splits, rmm::mr::get_default_resource());
   
   //verify_split_results(t, out, splits, true);           
   
   size_t num_out_tables = out.size();
   for(size_t idx=0; idx<num_out_tables; idx++){
      print_table(out[idx].table);
   }   

   int whee = 10;
   whee++; 
   */  

   /*         
   int num_els = 3;
   int c0[] = { 0, 1, 2 };
   wrapper<int> c0_w(c0, c0 + num_els);
   std::vector<size_type> splits { 0, 2, 1, 3 };
      
   auto out = cudf::experimental::slice(c0_w, splits);

   for(size_t idx=0; idx<out.size(); idx++){
      print_column(out[idx]);      
   }
   */     
   
   /*
   auto c = make_strings( {"1", "2", "3", "4", "5" } );
   column_view cv = c->view();   

   std::vector<size_type> ssplits { 1, 5 };
   auto sout = cudf::experimental::slice(cv, ssplits);
   
   for(size_t idx=0; idx<sout.size(); idx++){
      print_column(sout[idx]);      
   } 

   int whee = 10;
   whee++;   
   */
              
    auto valids = cudf::test::make_counting_transform_iterator(0, [](auto i) { return i%2 == 0 ? true : false; });   
    std::vector<std::string> strings[2]     = { {"this", "is", "a", "column", "of", "strings"}, 
                                                {"one", "two", "three", "four", "five", "six"} };
    cudf::test::strings_column_wrapper sw[2] = { {strings[0].begin(), strings[0].end(), valids},
                                                 {strings[1].begin(), strings[1].end(), valids} };

    std::vector<std::unique_ptr<cudf::column>> scols;
    scols.push_back(sw[0].release());
    scols.push_back(sw[1].release());
    cudf::experimental::table src_table(std::move(scols));

    std::vector<cudf::size_type> splits{2};
    
    std::vector<cudf::experimental::table> expected = create_expected_string_tables_for_splits(strings, splits, true);

    auto result = cudf::experimental::contiguous_split(src_table, splits, rmm::mr::get_default_resource());
    
    EXPECT_EQ(expected.size(), result.size());

    for (unsigned long index = 0; index < result.size(); index++) {       
        {
         
         printf("---------------------\n");
         print_table(expected[index]);
         print_table(result[index].table);
         printf("---------------------\n");
         cudf::test::expect_tables_equal(expected[index], result[index].table);
      }
    } 
    

   /*
    auto valids = cudf::test::make_counting_transform_iterator(0, [](auto i) { return true; });
    std::vector<std::string> strings = { "this", "is", "a", "column", "of", "strings", "with", "in", "valid" };
    cudf::test::strings_column_wrapper sw = { strings.begin(), strings.end(), valids };
    auto scols = sw.release();    
   
    std::vector<cudf::size_type> splits{2, 5, 9};        

    std::vector<cudf::column_view> result = cudf::experimental::split(*scols, splits);

   size_t idx;
   for(idx=0; idx<result.size(); idx++){
      cudf::test::print(result[idx]); std::cout << "\n";      
   }
   */
}