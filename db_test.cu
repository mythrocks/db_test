#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <unistd.h>

#include <cudf/cudf.h>
#include <tests/utilities/legacy/column_wrapper.cuh>
#include <bitmask/legacy/bit_mask.cuh>
#include <bitmask/legacy/legacy_bitmask.hpp>
#include <bitmask/legacy/bit_mask.cuh>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <cudf/copying.hpp>
#include <cudf/replace.hpp>
#include <utilities/cuda_utils.hpp>

#include <tests/utilities/column_wrapper.hpp>
#include <tests/utilities/column_utilities.hpp>

// ------------------
//
// Defines
//
// ------------------

#define BLOCK_SIZE           (256)

#define UNREFERENCED(_x)    do { (void)(_x); } while(0)

struct scoped_timer {
    timespec m_start;
    char m_name[64];

    scoped_timer(){}
    scoped_timer(const char *_name) 
    {         
        strcpy(m_name, _name);
        clock_gettime(CLOCK_MONOTONIC, &m_start);
    }
    ~scoped_timer()
    {
        timespec end;
        clock_gettime(CLOCK_MONOTONIC, &end);    
        long total = ((1000000000 * end.tv_sec) + end.tv_nsec) - 
                     ((1000000000 * m_start.tv_sec) + m_start.tv_nsec);      
        printf("%s : %.2f us\n", m_name, (float)total / (float)1000000.0f);
    }
};

#if 0    // skeleton for working in cudf
namespace db_test {

using namespace cudf;
using namespace std;
using namespace rmm;
using namespace rmm::mr;

namespace {    // anonymous
   // functor and lambdas go here
}  // end anonymous namespace

namespace cudf {
namespace detail {
   // detail/internal versions of exposed functions go here
}  // namespace detail

   // externally exposed functions go here
}  // namespace cudf

} 
#endif   // skeleton for working in cudf

// ------------------
//
// Internal functions
//
// ------------------

// there's some "do stuff the first time" issues that cause bogus timings.
// this function just flushes all that junk out
static void clear_baffles()
{
    // doing an alloc, a memcpy and a free seems to do the trick
    void *gpu_data;
    rmmError_t err = RMM_ALLOC(&gpu_data, 8 * 1024 * 1024, 0);    
    uint cpu_data[64] = { 0 };
    hipMemcpy(gpu_data, cpu_data, sizeof(cpu_data), hipMemcpyHostToDevice);    
    RMM_FREE(gpu_data, 0);

    // can't hurt if there's some weird async stuff happening
    sleep(1);
}

#if 0 // sort a column using thrust::sort
// sort a column directly using thrust::sort
static void sort_column_basic()
{   
    int idx;

    // some source data.
    int num_rows = 16;
    float cpu_data[16] = { 5, 8, 10, 11, 2, 3, 1, 15, 12, 7, 6, 13, 9, 4, 0, 14 };
    printf("Unsorted: ");
    for(idx=0; idx<num_rows; idx++){        
        printf(idx < num_rows ? "%.2f, " : "%.2f", cpu_data[idx]);
    }    
    printf("\n");

    int data_size = num_rows * sizeof(float);  

    // allocate device memory for the floats
    float *gpu_data = nullptr;        
    rmmError_t err = RMM_ALLOC(&gpu_data, data_size, 0);    

    // copy cpu data over        
    hipError_t mem_err = hipMemcpy(gpu_data, cpu_data, data_size, hipMemcpyHostToDevice);    

    // setup the column struct. validity mask is null indicating "everything is valid"
    //gdf_column gpu_column;
    //gdf_column_view(&gpu_column, gpu_data, nullptr, num_rows, GDF_FLOAT32);

    // sort
    thrust::device_ptr<float> dv(gpu_data);
    thrust::sort(dv, dv + num_rows, thrust::less<float>());

    // grab the data back
    hipMemcpy(cpu_data, gpu_data, data_size, hipMemcpyDeviceToHost);        

    printf("Sorted: ");
    for(idx=0; idx<num_rows; idx++){        
        printf(idx < num_rows ? "%.2f, " : "%.2f", cpu_data[idx]);
    }    
    printf("\n\n");

    RMM_FREE(gpu_data, 0);    
}
#endif // sort a column using thrust::sort

#if 0 // old normalize_nans_and_zeros kernel method. never got used.
namespace db_test {

using namespace cudf;
using namespace std;
using namespace rmm;
using namespace rmm::mr;

// old normalize_nans_and_zeros kernel method. never got used.
namespace {  // anonymous

/* --------------------------------------------------------------------------*/
/**
 * @brief Kernel that converts inputs from `in` to `out`  using the following
 *        rule:   Convert  -NaN  -> NaN
 *                Convert  -0.0  -> 0.0
 *
 * @param[in] column_device_view representing input data
 * @param[in] mutable_column_device_view representing output data. can be
 *            the same actual underlying buffer that in points to. 
 *
 * @returns
 */
/* ----------------------------------------------------------------------------*/
template <typename T>
__global__
void normalize_nans_and_zeros(column_device_view in, 
                              mutable_column_device_view out)
{
   int tid = threadIdx.x;
   int blkid = blockIdx.x;
   int blksz = blockDim.x;
   int gridsz = gridDim.x;

   int start = tid + blkid * blksz;
   int step = blksz * gridsz;

   // grid-stride
   for (int i=start; i<in.size(); i+=step) {
      if(!in.is_valid(i)){
         continue;
      }

      T el = in.element<T>(i);
      if(std::isnan(el)){
         out.element<T>(i) = std::numeric_limits<T>::quiet_NaN();
      } else if(el == (T)-0.0){
         out.element<T>(i) = (T)0.0;
      } else {
         out.element<T>(i) = el;
      }
   }
}                        

  /* --------------------------------------------------------------------------*/
  /**
   * @brief Functor called by the `type_dispatcher` in order to invoke and instantiate
   *        `normalize_nans_and_zeros` with the appropriate data types.
   */
  /* ----------------------------------------------------------------------------*/
struct normalize_nans_and_zeros_kernel_forwarder {
   // floats and doubles. what we really care about.
   template <typename T, std::enable_if_t<std::is_floating_point<T>::value>* = nullptr>
   void operator()(  column_device_view in,
                     mutable_column_device_view out,
                     hipStream_t stream)
   {
      cudf::util::cuda::grid_config_1d grid{in.size(), BLOCK_SIZE};
      normalize_nans_and_zeros<T><<<grid.num_blocks, BLOCK_SIZE, 0, stream>>>(in, out);
   }

   // if we get in here for anything but a float or double, that's a problem.
   template <typename T, std::enable_if_t<not std::is_floating_point<T>::value>* = nullptr>
   void operator()(  column_device_view in,
                     mutable_column_device_view out,
                     hipStream_t stream)
   {
      CUDF_FAIL("Unexpected non floating-point type.");      
   }   
};

} // end anonymous namespace

namespace cudf {
namespace detail {

std::unique_ptr<column> normalize_nans_and_zeros( column_view input,                                                  
                                                  hipStream_t stream,
                                                  rmm::mr::device_memory_resource *mr = rmm::mr::get_default_resource())
{   
    // to device. unique_ptr which gets automatically cleaned up when we leave
   auto device_in = column_device_view::create(input);
   
   // ultimately, the output.
   auto out = make_numeric_column(input.type(), input.size(), ALL_VALID, stream);
   // from device. unique_ptr which gets automatically cleaned up when we leave.
   auto device_out = mutable_column_device_view::create(*out);

   // invoke the actual kernel.  
   experimental::type_dispatcher(input.type(), 
                                 normalize_nans_and_zeros_kernel_forwarder{},
                                 *device_in,
                                 *device_out,
                                 stream);

   return out;                 
}                                                 

void normalize_nans_and_zeros(mutable_column_view in_out,
                              hipStream_t stream)
{  
   // wrapping the in_out data in a column_view so we can call the same lower level code.
   // that we use for the non in-place version.
   column_view input = in_out;

   // to device. unique_ptr which gets automatically cleaned up when we leave
   auto device_in = column_device_view::create(input);

   // from device. unique_ptr which gets automatically cleaned up when we leave.   
   auto device_out = mutable_column_device_view::create(in_out);

    // invoke the actual kernel.  
   experimental::type_dispatcher(input.type(), 
                                 normalize_nans_and_zeros_kernel_forwarder{},
                                 *device_in,
                                 *device_out,
                                 stream);
} 

}  // namespace detail

/**
 * @brief Function that converts inputs from `input` using the following rule
 *        rule:   Convert  -NaN  -> NaN
 *                Convert  -0.0  -> 0.0
 *
 * @param[in] column_device_view representing input data
 * @param[in] device_memory_resource allocator for allocating output data 
 *
 * @returns new column
 */
std::unique_ptr<column> normalize_nans_and_zeros( column_view input,                                                                                                    
                                                  rmm::mr::device_memory_resource *mr = rmm::mr::get_default_resource())
{
   return detail::normalize_nans_and_zeros(input, 0, mr);;
}

/**
 * @brief Function that processes values in-place from `in_out` using the following rule
 *        rule:   Convert  -NaN  -> NaN
 *                Convert  -0.0  -> 0.0
 *
 * @param[in, out] mutable_column_view representing input data. data is processed in-place
 *
 * @returns new column
 */
void normalize_nans_and_zeros(mutable_column_view in_out)
{
   return detail::normalize_nans_and_zeros(in_out, 0);
}

} // namespace cudf

} // anonymous namespace

void ntest()
{
   float whee[10] = { 32.5f, -0.0f, 111.0f, -NAN, NAN, 1.0f, 0.0f, 54.3f };   
   int num_els = 8;

   uint32_t nan = *((uint32_t*)(&whee[1]));   

   printf("Before: ");
   for(int idx=0; idx<num_els; idx++){
      printf(idx < num_els ? "%.2f, " : "%.2f", whee[idx]);
   }
   printf("\n");

   // copy the data to a column (which is always on the device)
   auto test_data = cudf::make_numeric_column(cudf::data_type(cudf::FLOAT32), num_els, cudf::ALL_VALID, 0);      
   // there's an overloaded operator for this but I like to see what's
   // actually going on.
   auto view = test_data->mutable_view();
   hipMemcpy(view.head(), whee, sizeof(float) * num_els, hipMemcpyHostToDevice);

   // do it
   db_test::cudf::normalize_nans_and_zeros(view);

   // get the data back
   hipMemcpy(whee, view.head(), sizeof(float) * num_els, hipMemcpyDeviceToHost);
   
   uint32_t nan2 = *((uint32_t*)(&whee[1]));

   printf("After: ");
   for(int idx=0; idx<num_els; idx++){
      printf(idx < num_els ? "%.2f, " : "%.2f", whee[idx]);
   }
   printf("\n\n");
}
#endif   // old normalize_nans_and_zeros kernel method. never got used.

#if 0    // copy_if_else
//namespace db_test {

using namespace cudf;
using namespace std;
using namespace rmm;
using namespace rmm::mr;

namespace {

using namespace cudf; 

/* --------------------------------------------------------------------------*/
/**
* @brief Functor called by the `type_dispatcher` in order to perform a copy if/else
*        using a filter function to select from lhs/rhs columns.
*/
/* ----------------------------------------------------------------------------*/
struct copy_if_else_functor {
   template <typename T, typename Filter>
   void operator()(  Filter filter,
                     column_view const& lhs,
                     column_view const& rhs,
                     mutable_column_view& out,
                     hipStream_t stream)
   {
      auto begin  = thrust::make_zip_iterator(thrust::make_tuple( thrust::make_counting_iterator(0),
                                                                  lhs.begin<T>(),
                                                                  rhs.begin<T>()));

      auto end  = thrust::make_zip_iterator(thrust::make_tuple(thrust::make_counting_iterator(lhs.size()),
                                                               lhs.end<T>(),
                                                               rhs.end<T>()));
      
      thrust::transform(rmm::exec_policy(stream)->on(stream), begin, end, out.begin<T>(),
                        [filter] __device__ (thrust::tuple<size_type, T, T> i)
                        {
                           return filter(thrust::get<0>(i)) ? thrust::get<1>(i) : thrust::get<2>(i);
                        });
   } 
};

}  // end anonymous namespace

namespace cudf {
namespace detail {

/**
 * @brief   Returns a new column, where each element is selected from either @p lhs or 
 *          @p rhs based on the filter lambda. 
 * 
 * @p filter must be a functor or lambda with the following signature:
 * __device__ bool operator()(cudf::size_type i);
 * It should return true if element i of @p lhs should be selected, or false if element i of @p rhs should be selected. 
 *         
 * @throws cudf::logic_error if lhs and rhs are not of the same type
 * @throws cudf::logic_error if lhs and rhs are not of the same length 
 * @param[in] filter lambda. 
 * @param[in] left-hand column_view
 * @param[in] right-hand column_view
 * @param[in] mr resource for allocating device memory
 *
 * @returns new column with the selected elements
 */
template<typename Filter>
unique_ptr<column> copy_if_else( Filter filter, column_view const& lhs, column_view const& rhs,
                                 rmm::mr::device_memory_resource *mr = rmm::mr::get_default_resource(),
                                 hipStream_t stream = 0)
{
   // output
   std::unique_ptr<column> out = experimental::allocate_like(lhs, lhs.size(), experimental::mask_allocation_policy::RETAIN, mr);
   auto mutable_view = out->mutable_view();
   
   cudf::experimental::type_dispatcher(lhs.type(), 
                                       copy_if_else_functor{},
                                       filter,
                                       lhs,
                                       rhs,
                                       mutable_view,
                                       stream);

   return out;
}

}  // namespace detail
}  // namespace cudf

namespace cudf {
namespace detail {

struct pfunk {
    column_device_view bool_mask_device;

    __device__ bool operator()(int i) const
    {
       return bool_mask_device.element<cudf::experimental::bool8>(i);
    }
};

unique_ptr<column> copy_if_else( column_view const& boolean_mask, column_view const& lhs, column_view const& rhs,
                                 rmm::mr::device_memory_resource *mr,
                                 hipStream_t stream)
{
   CUDF_EXPECTS(lhs.type() == rhs.type(), "Both columns must be of the same type");
   CUDF_EXPECTS(lhs.size() == rhs.size(), "Both columns must be of the size");
   CUDF_EXPECTS(boolean_mask.type() == data_type(BOOL8), "Boolean mask column must be of type BOOL8");   
   CUDF_EXPECTS(boolean_mask.size() == lhs.size(), "Boolean mask column must be the same size as lhs and rhs columns");

   // filter in this case is a column made of bools
   auto bool_mask_device_ptr = column_device_view::create(boolean_mask);   
   column_device_view bool_mask_device = *bool_mask_device_ptr;   
   auto filter = [bool_mask_device] __device__ (cudf::size_type i) { return bool_mask_device.element<cudf::experimental::bool8>(i); }; 

   return copy_if_else(filter, lhs, rhs, mr, stream);
}

}  // namespace detail

/**
 * @brief   Returns a new column, where each element is selected from either @p lhs or 
 *          @p rhs based on the value of the corresponding element in @p boolean_mask
 *
 * Selects each element i in the output column from either @p rhs or @p lhs using the following rule:
 *          output[i] = (boolean_mask[i] == true) ? lhs[i] : rhs[i]
 *         
 * @throws cudf::logic_error if lhs and rhs are not of the same type
 * @throws cudf::logic_error if lhs and rhs are not of the same length
 * @throws cudf::logic_error if boolean mask is not of type bool8
 * @throws cudf::logic_error if boolean mask is not of the same length as lhs and rhs 
 * @param[in] column_view representing "left (true) / right (false)" boolean for each element
 * @param[in] left-hand column_view
 * @param[in] right-hand column_view
 * @param[in] mr resource for allocating device memory
 *
 * @returns new column with the selected elements
 */
unique_ptr<column> copy_if_else( column_view const& boolean_mask, column_view const& lhs, column_view const& rhs, 
                                 rmm::mr::device_memory_resource *mr = rmm::mr::get_default_resource())
{
   return detail::copy_if_else(boolean_mask, lhs, rhs, mr, 0);
}

}  // namespace cudf


// to keep names shorter
#define wrapper cudf::test::fixed_width_column_wrapper
using bool_wrapper = wrapper<cudf::experimental::bool8>;

template<typename T>
void copy_if_else_check(bool_wrapper const&  mask_w,
                        wrapper<T> const&    lhs_w,
                        wrapper<T> const&    rhs_w,
                        wrapper<T> const&    expected_w)
{
   // construct input views
   column mask(mask_w);
   column_view mask_v(mask);
   //
   column lhs(lhs_w);
   column_view lhs_v = lhs.view();
   //
   column rhs(rhs_w);
   column_view rhs_v = rhs.view();
   //
   column expected(expected_w);
   column_view expected_v = expected.view();

   // get the result
   auto out = cudf::copy_if_else(mask_v, lhs_v, rhs_v);
   column_view out_v = out->view();   

   T whee[64];
   hipMemcpy(whee, out_v.head(), sizeof(T) * out_v.size(), hipMemcpyDeviceToHost);

   // compare
   cudf::test::expect_columns_equal(out_v, expected_v);
}

void copy_if_else_test()
{
   {
      bool_wrapper   mask_w      { true, true, false, true, true }; 
      wrapper<int>   lhs_w       { 5, 5, 5, 5, 5 };
      wrapper<int>   rhs_w       { 6, 6, 6, 6, 6 };
      wrapper<int>   expected_w  { 5, 5, 6, 5, 5 };
      copy_if_else_check(mask_w, lhs_w, rhs_w, expected_w); 
   }

   {
      bool_wrapper   mask_w      { false, true, false, false, true };
      wrapper<double>lhs_w       { -10.0f, -10.0, -10.0, -10.0, -10.0 };
      wrapper<double>rhs_w       { 7.0, 7.0, 7.0, 7.0, 7.0 };
      wrapper<double>expected_w  { 7.0, -10.0, 7.0, 7.0, -10.0 };
      copy_if_else_check(mask_w, lhs_w, rhs_w, expected_w);
   }
}

#endif   // copy if else

//}  // db_test

int main()
{                
   // init stuff
   hipInit(0);    
   rmmOptions_t rmm{};
   rmm.allocation_mode = CudaDefaultAllocation;
   rmm.initial_pool_size = 16 * 1024 * 1024;
   rmm.enable_logging = false;
   rmmInitialize(&rmm);      

   // there's some "do stuff the first time" issues that cause bogus timings.
   // this function just flushes all that junk out
   clear_baffles();   

    // shut stuff down
   rmmFinalize();
}
