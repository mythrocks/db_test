#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <unistd.h>

#include <cudf/cudf.h>
#include <tests/utilities/legacy/column_wrapper.cuh>
#include <bitmask/legacy/bit_mask.cuh>
#include <bitmask/legacy/legacy_bitmask.hpp>
#include <bitmask/legacy/bit_mask.cuh>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <cudf/replace.hpp>
#include <utilities/cuda_utils.hpp>

#include <tests/utilities/column_wrapper.hpp>
#include <tests/utilities/column_utilities.hpp>

// ------------------
//
// Defines
//
// ------------------

#define BLOCK_SIZE           (256)

#define UNREFERENCED(_x)    do { (void)(_x); } while(0)

struct scoped_timer {
    timespec m_start;
    char m_name[64];

    scoped_timer(){}
    scoped_timer(const char *_name) 
    {         
        strcpy(m_name, _name);
        clock_gettime(CLOCK_MONOTONIC, &m_start);
    }
    ~scoped_timer()
    {
        timespec end;
        clock_gettime(CLOCK_MONOTONIC, &end);    
        long total = ((1000000000 * end.tv_sec) + end.tv_nsec) - 
                     ((1000000000 * m_start.tv_sec) + m_start.tv_nsec);      
        printf("%s : %.2f us\n", m_name, (float)total / (float)1000000.0f);
    }
};

#if 0    // skeleton for working in cudf
namespace db_test {

using namespace cudf;
using namespace std;
using namespace rmm;
using namespace rmm::mr;

} 
#endif   // skeleton for working in cudf

// ------------------
//
// Internal functions
//
// ------------------

// there's some "do stuff the first time" issues that cause bogus timings.
// this function just flushes all that junk out
static void clear_baffles()
{
    // doing an alloc, a memcpy and a free seems to do the trick
    void *gpu_data;
    rmmError_t err = RMM_ALLOC(&gpu_data, 8 * 1024 * 1024, 0);    
    uint cpu_data[64] = { 0 };
    hipMemcpy(gpu_data, cpu_data, sizeof(cpu_data), hipMemcpyHostToDevice);    
    RMM_FREE(gpu_data, 0);

    // can't hurt if there's some weird async stuff happening
    sleep(1);
}

#if 0 // sort a column using thrust::sort
// sort a column directly using thrust::sort
static void sort_column_basic()
{   
    int idx;

    // some source data.
    int num_rows = 16;
    float cpu_data[16] = { 5, 8, 10, 11, 2, 3, 1, 15, 12, 7, 6, 13, 9, 4, 0, 14 };
    printf("Unsorted: ");
    for(idx=0; idx<num_rows; idx++){        
        printf(idx < num_rows ? "%.2f, " : "%.2f", cpu_data[idx]);
    }    
    printf("\n");

    int data_size = num_rows * sizeof(float);  

    // allocate device memory for the floats
    float *gpu_data = nullptr;        
    rmmError_t err = RMM_ALLOC(&gpu_data, data_size, 0);    

    // copy cpu data over        
    hipError_t mem_err = hipMemcpy(gpu_data, cpu_data, data_size, hipMemcpyHostToDevice);    

    // setup the column struct. validity mask is null indicating "everything is valid"
    //gdf_column gpu_column;
    //gdf_column_view(&gpu_column, gpu_data, nullptr, num_rows, GDF_FLOAT32);

    // sort
    thrust::device_ptr<float> dv(gpu_data);
    thrust::sort(dv, dv + num_rows, thrust::less<float>());

    // grab the data back
    hipMemcpy(cpu_data, gpu_data, data_size, hipMemcpyDeviceToHost);        

    printf("Sorted: ");
    for(idx=0; idx<num_rows; idx++){        
        printf(idx < num_rows ? "%.2f, " : "%.2f", cpu_data[idx]);
    }    
    printf("\n\n");

    RMM_FREE(gpu_data, 0);    
}
#endif // sort a column using thrust::sort

#if 0 // old normalize_nans_and_zeros kernel method. never got used.
namespace db_test {

using namespace cudf;
using namespace std;
using namespace rmm;
using namespace rmm::mr;

// old normalize_nans_and_zeros kernel method. never got used.
namespace {  // anonymous

/* --------------------------------------------------------------------------*/
/**
 * @brief Kernel that converts inputs from `in` to `out`  using the following
 *        rule:   Convert  -NaN  -> NaN
 *                Convert  -0.0  -> 0.0
 *
 * @param[in] column_device_view representing input data
 * @param[in] mutable_column_device_view representing output data. can be
 *            the same actual underlying buffer that in points to. 
 *
 * @returns
 */
/* ----------------------------------------------------------------------------*/
template <typename T>
__global__
void normalize_nans_and_zeros(column_device_view in, 
                              mutable_column_device_view out)
{
   int tid = threadIdx.x;
   int blkid = blockIdx.x;
   int blksz = blockDim.x;
   int gridsz = gridDim.x;

   int start = tid + blkid * blksz;
   int step = blksz * gridsz;

   // grid-stride
   for (int i=start; i<in.size(); i+=step) {
      if(!in.is_valid(i)){
         continue;
      }

      T el = in.element<T>(i);
      if(std::isnan(el)){
         out.element<T>(i) = std::numeric_limits<T>::quiet_NaN();
      } else if(el == (T)-0.0){
         out.element<T>(i) = (T)0.0;
      } else {
         out.element<T>(i) = el;
      }
   }
}                        

  /* --------------------------------------------------------------------------*/
  /**
   * @brief Functor called by the `type_dispatcher` in order to invoke and instantiate
   *        `normalize_nans_and_zeros` with the appropriate data types.
   */
  /* ----------------------------------------------------------------------------*/
struct normalize_nans_and_zeros_kernel_forwarder {
   // floats and doubles. what we really care about.
   template <typename T, std::enable_if_t<std::is_floating_point<T>::value>* = nullptr>
   void operator()(  column_device_view in,
                     mutable_column_device_view out,
                     hipStream_t stream)
   {
      cudf::util::cuda::grid_config_1d grid{in.size(), BLOCK_SIZE};
      normalize_nans_and_zeros<T><<<grid.num_blocks, BLOCK_SIZE, 0, stream>>>(in, out);
   }

   // if we get in here for anything but a float or double, that's a problem.
   template <typename T, std::enable_if_t<not std::is_floating_point<T>::value>* = nullptr>
   void operator()(  column_device_view in,
                     mutable_column_device_view out,
                     hipStream_t stream)
   {
      CUDF_FAIL("Unexpected non floating-point type.");      
   }   
};

} // end anonymous namespace

namespace cudf {
namespace detail {

std::unique_ptr<column> normalize_nans_and_zeros( column_view input,                                                  
                                                  hipStream_t stream,
                                                  rmm::mr::device_memory_resource *mr = rmm::mr::get_default_resource())
{   
    // to device. unique_ptr which gets automatically cleaned up when we leave
   auto device_in = column_device_view::create(input);
   
   // ultimately, the output.
   auto out = make_numeric_column(input.type(), input.size(), ALL_VALID, stream);
   // from device. unique_ptr which gets automatically cleaned up when we leave.
   auto device_out = mutable_column_device_view::create(*out);

   // invoke the actual kernel.  
   experimental::type_dispatcher(input.type(), 
                                 normalize_nans_and_zeros_kernel_forwarder{},
                                 *device_in,
                                 *device_out,
                                 stream);

   return out;                 
}                                                 

void normalize_nans_and_zeros(mutable_column_view in_out,
                              hipStream_t stream)
{  
   // wrapping the in_out data in a column_view so we can call the same lower level code.
   // that we use for the non in-place version.
   column_view input = in_out;

   // to device. unique_ptr which gets automatically cleaned up when we leave
   auto device_in = column_device_view::create(input);

   // from device. unique_ptr which gets automatically cleaned up when we leave.   
   auto device_out = mutable_column_device_view::create(in_out);

    // invoke the actual kernel.  
   experimental::type_dispatcher(input.type(), 
                                 normalize_nans_and_zeros_kernel_forwarder{},
                                 *device_in,
                                 *device_out,
                                 stream);
} 

}  // namespace detail

/**
 * @brief Function that converts inputs from `input` using the following rule
 *        rule:   Convert  -NaN  -> NaN
 *                Convert  -0.0  -> 0.0
 *
 * @param[in] column_device_view representing input data
 * @param[in] device_memory_resource allocator for allocating output data 
 *
 * @returns new column
 */
std::unique_ptr<column> normalize_nans_and_zeros( column_view input,                                                                                                    
                                                  rmm::mr::device_memory_resource *mr = rmm::mr::get_default_resource())
{
   return detail::normalize_nans_and_zeros(input, 0, mr);;
}

/**
 * @brief Function that processes values in-place from `in_out` using the following rule
 *        rule:   Convert  -NaN  -> NaN
 *                Convert  -0.0  -> 0.0
 *
 * @param[in, out] mutable_column_view representing input data. data is processed in-place
 *
 * @returns new column
 */
void normalize_nans_and_zeros(mutable_column_view in_out)
{
   return detail::normalize_nans_and_zeros(in_out, 0);
}

} // namespace cudf

} // anonymous namespace

void ntest()
{
   float whee[10] = { 32.5f, -0.0f, 111.0f, -NAN, NAN, 1.0f, 0.0f, 54.3f };   
   int num_els = 8;

   uint32_t nan = *((uint32_t*)(&whee[1]));   

   printf("Before: ");
   for(int idx=0; idx<num_els; idx++){
      printf(idx < num_els ? "%.2f, " : "%.2f", whee[idx]);
   }
   printf("\n");

   // copy the data to a column (which is always on the device)
   auto test_data = cudf::make_numeric_column(cudf::data_type(cudf::FLOAT32), num_els, cudf::ALL_VALID, 0);      
   // there's an overloaded operator for this but I like to see what's
   // actually going on.
   auto view = test_data->mutable_view();
   hipMemcpy(view.head(), whee, sizeof(float) * num_els, hipMemcpyHostToDevice);

   // do it
   db_test::cudf::normalize_nans_and_zeros(view);

   // get the data back
   hipMemcpy(whee, view.head(), sizeof(float) * num_els, hipMemcpyDeviceToHost);
   
   uint32_t nan2 = *((uint32_t*)(&whee[1]));

   printf("After: ");
   for(int idx=0; idx<num_els; idx++){
      printf(idx < num_els ? "%.2f, " : "%.2f", whee[idx]);
   }
   printf("\n\n");
}
#endif   // old normalize_nans_and_zeros kernel method. never got used.

int main()
{                
    // init stuff
    hipInit(0);    
    rmmOptions_t rmm{};
    rmm.allocation_mode = CudaDefaultAllocation;
    rmm.initial_pool_size = 16 * 1024 * 1024;
    rmm.enable_logging = false;
    rmmInitialize(&rmm);      

    // there's some "do stuff the first time" issues that cause bogus timings.
    // this function just flushes all that junk out
    clear_baffles();    

    // shut stuff down
    rmmFinalize();
}
